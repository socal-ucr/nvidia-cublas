/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the CUBLAS helper functions */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

/* the next two macro definitions trigger
 * code generation when tlsHook.h is included
 */ 
#define __tlsHookIdentifier cublasThreadContext
#define __tlsHookType       struct cublasContext
#include <tlshook.h>

void cublasSetError (struct cublasContext *ctx, hipblasStatus_t error)
{
    if (ctx) {
        ctx->cublasLastError = error;
    }
}

int cublasInitialized (const struct cublasContext *ctx)
{
    if (!ctx) {
        return 0;
    } else {
        return ctx->cublasIsInitialized;
    }
}

/* 
 * For a given vector size, cublasVectorSplay() determines what CTA grid 
 * size to use, and how many threads per CTA.
 */
void cublasVectorSplay (int n, int tMin, int tMax, int gridW, int *nbrCtas, 
                        int *elemsPerCta, int *threadsPerCta)
{
    if (n < tMin) {
        *nbrCtas = 1;
        *elemsPerCta = n;
        *threadsPerCta = tMin;
    } else if (n < (gridW * tMin)) {
        *nbrCtas = ((n + tMin - 1) / tMin);
        *threadsPerCta = tMin;
        *elemsPerCta = *threadsPerCta;
    } else if (n < (gridW * tMax)) {
        int grp;
        *nbrCtas = gridW;
        grp = ((n + tMin - 1) / tMin);
        *threadsPerCta = (((grp + gridW -1) / gridW) * tMin);
        *elemsPerCta = *threadsPerCta;
    } else {
        int grp;
        *nbrCtas = gridW;
        *threadsPerCta = tMax;
        grp = ((n + tMin - 1) / tMin);
        grp = ((grp + gridW - 1) / gridW);
        *elemsPerCta = grp * tMin;
    }
}

void cublasShutDownCtx (struct cublasContext *ctx)
{
}

__tlsHookStatus cublasInitCtx (struct cublasContext *ctx, void *_status)
{
    hipblasStatus_t* status = (hipblasStatus_t*)_status;
    
    if (!ctx) {
        if (status) *status = HIPBLAS_STATUS_ALLOC_FAILED;
        return __tlsHookStatusFAIL;
    }
    ctx->cublasIsInitialized = false;
    ctx->cublasLastError = HIPBLAS_STATUS_SUCCESS;
    
    /* This will do nothing really but will initialize CUDA as a side effect */
    if (hipFree ((void *)0) != hipSuccess) {
        if (status) *status = HIPBLAS_STATUS_NOT_INITIALIZED;
        return __tlsHookStatusFAIL;
    }
    ctx->cublasIsInitialized = true;
    if (status) {
        *status = HIPBLAS_STATUS_SUCCESS;
    }
    
    return __tlsHookStatusOK;
}

/* --------------------------- CUBLAS API functions ------------------------ */

/*
 * hipblasStatus_t 
 * cublasInit (void)
 *
 * initializes the CUBLAS library and must be called before any other 
 * CUBLAS API function is invoked. It allocates hardware resources 
 * necessary for accessing the GPU.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_ALLOC_FAILED     if resources could not be allocated
 * HIPBLAS_STATUS_SUCCESS          if CUBLAS library initialized successfully
 */
hipblasStatus_t CUBLASAPI cublasInit (void)
{
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    
    (void)__tlsHookInitTlsValueForcublasThreadContext(cublasInitCtx, 
                                                      cublasShutDownCtx, 
                                                      &status);
    return status;
}

/*
 * hipblasStatus_t 
 * cublasShutdown (void)
 *
 * releases CPU-side resources used by the CUBLAS library. The release of 
 * GPU-side resources may be deferred until the application shuts down.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_SUCCESS          if CUBLAS library shut down successfully
 */
hipblasStatus_t CUBLASAPI cublasShutdown (void)
{
    __tlsHookClearTlsValue(&cublasThreadContext);
    return HIPBLAS_STATUS_SUCCESS;
}

/* 
 * hipblasStatus_t 
 * cublasGetError (void)
 *
 * returns the last error that occurred on invocation of any of the
 * CUBLAS BLAS functions. While the CUBLAS helper functions return status
 * directly, the BLAS functions do not do so for improved 
 * compatibility with existing environments that do not expect BLAS
 * functions to return status. Reading the error status via 
 * cublasGetError() resets the internal error state to 
 * HIPBLAS_STATUS_SUCCESS.
 */
hipblasStatus_t CUBLASAPI cublasGetError (void)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    if (!ctx) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    } else {
        hipblasStatus_t retVal = ctx->cublasLastError;
        ctx->cublasLastError = HIPBLAS_STATUS_SUCCESS;
        return retVal;
    }
}

/*
 * hipblasStatus_t 
 * cublasAlloc (int n, int elemSize, void **devicePtr)
 *
 * creates an object in GPU memory space capable of holding an array of
 * n elements, where each element requires elemSize bytes of storage. If 
 * the function call is successful, a pointer to the object in GPU memory 
 * space is placed in devicePtr. Note that this is a device pointer that
 * cannot be dereferenced in host code.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n <= 0, or elemSize <= 0
 * HIPBLAS_STATUS_ALLOC_FAILED     if the object could not be allocated due to
 *                                lack of resources.
 * HIPBLAS_STATUS_SUCCESS          if storage was successfully allocated
 */
hipblasStatus_t CUBLASAPI cublasAlloc (int n, int elemSize, void **devicePtr)
{
    hipError_t cudaStat;
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    *devicePtr = 0;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if ((n <= 0) || (elemSize <= 0)) {        
        return HIPBLAS_STATUS_INVALID_VALUE;
    }
    cudaStat = hipMalloc (devicePtr, elemSize * n);
    if (cudaStat != hipSuccess) {
        return HIPBLAS_STATUS_ALLOC_FAILED;
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/*
 * hipblasStatus_t 
 * cublasFree (const void *devicePtr)
 *
 * destroys the object in GPU memory space pointed to by devicePtr.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INTERNAL_ERROR   if the object could not be deallocated
 * HIPBLAS_STATUS_SUCCESS          if object was destroyed successfully
 */
hipblasStatus_t CUBLASAPI cublasFree (const void *devicePtr)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    hipError_t cudaStat;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if (devicePtr) {
        cudaStat = hipFree ((void *)devicePtr);
        if (cudaStat != hipSuccess) {
            /* should never fail, except when there is internal corruption*/
            return HIPBLAS_STATUS_INTERNAL_ERROR;
        }
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/* 
 * hipblasStatus_t 
 * hipblasSetVector (int n, int elemSize, const void *x, int incx, 
 *                  void *y, int incy) 
 *
 * copies n elements from a vector x in CPU memory space to a vector y 
 * in GPU memory space. Elements in both vectors are assumed to have a 
 * size of elemSize bytes. Storage spacing between consecutive elements
 * is incx for the source vector x and incy for the destination vector
 * y. In general, y points to an object, or part of an object, allocated
 * via cublasAlloc(). Column major format for two-dimensional matrices
 * is assumed throughout CUBLAS. Therefore, if the increment for a vector 
 * is equal to 1, this access a column vector while using an increment 
 * equal to the leading dimension of the respective matrix accesses a 
 * row vector.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx, incy, or elemSize <= 0
 * HIPBLAS_STATUS_MAPPING_ERROR    if an error occurred accessing GPU memory   
 * HIPBLAS_STATUS_SUCCESS          if the operation completed successfully
 */
hipblasStatus_t CUBLASAPI hipblasSetVector (int n, int elemSize, 
                                        const void *hostPtr, int incx, 
                                        void *devicePtr, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    hipError_t cudaStat = hipSuccess;
    const char *sp = (const char *)hostPtr;
    char *dp = (char *)devicePtr;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if ((incx <= 0) || (incy <= 0) || (elemSize <= 0)) {
        return HIPBLAS_STATUS_INVALID_VALUE;
    }
    /* early out if nothing to do */
    if (n == 0) {
        return HIPBLAS_STATUS_SUCCESS;
    }
    
    if ((incx == 1) && (incy == 1)) {
        cudaStat = hipMemcpy (dp, sp, n * elemSize, hipMemcpyHostToDevice);
    } else {
        cudaStat = hipMemcpy2D (dp, incy * elemSize, sp, incx * elemSize,
                                 elemSize, n, hipMemcpyHostToDevice);
    }
    if (cudaStat != hipSuccess) {
        return HIPBLAS_STATUS_MAPPING_ERROR;
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/* 
 * hipblasStatus_t 
 * hipblasGetVector (int n, int elemSize, const void *x, int incx, 
 *                  void *y, int incy)
 * 
 * copies n elements from a vector x in GPU memory space to a vector y 
 * in CPU memory space. Elements in both vectors are assumed to have a 
 * size of elemSize bytes. Storage spacing between consecutive elements
 * is incx for the source vector x and incy for the destination vector
 * y. In general, x points to an object, or part of an object, allocated
 * via cublasAlloc(). Column major format for two-dimensional matrices
 * is assumed throughout CUBLAS. Therefore, if the increment for a vector 
 * is equal to 1, this access a column vector while using an increment 
 * equal to the leading dimension of the respective matrix accesses a 
 * row vector.
 *
 * Return Values
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx, incy, or elemSize <= 0
 * HIPBLAS_STATUS_MAPPING_ERROR    if an error occurred accessing GPU memory   
 * HIPBLAS_STATUS_SUCCESS          if the operation completed successfully
 */
hipblasStatus_t CUBLASAPI hipblasGetVector (int n, int elemSize,
                                        const void *devicePtr, int incx,
                                        void *hostPtr, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    hipError_t cudaStat = hipSuccess;
    const char *sp = (const char *)devicePtr;
    char *dp = (char *)hostPtr;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if ((incx <= 0) || (incy <= 0) || (elemSize <= 0)) {
        return HIPBLAS_STATUS_INVALID_VALUE;
    }
    /* early out if nothing to do */
    if (n == 0) {
        return HIPBLAS_STATUS_SUCCESS;
    }
    if ((incx == 1) && (incy == 1)) {
        cudaStat = hipMemcpy (dp, sp, n * elemSize, hipMemcpyDeviceToHost);
    } else {
        cudaStat = hipMemcpy2D (dp, incy * elemSize, sp, incx * elemSize,
                                 elemSize, n, hipMemcpyDeviceToHost);
    }
    if (cudaStat != hipSuccess) {
        return HIPBLAS_STATUS_MAPPING_ERROR;
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/*
 * hipblasStatus_t 
 * hipblasSetMatrix (int rows, int cols, int elemSize, const void *A, 
 *                  int lda, void *B, int ldb)
 *
 * copies a tile of rows x cols elements from a matrix A in CPU memory
 * space to a matrix B in GPU memory space. Each element requires storage
 * of elemSize bytes. Both matrices are assumed to be stored in column 
 * major format, with the leading dimension (i.e. number of rows) of 
 * source matrix A provided in lda, and the leading dimension of matrix B
 * provided in ldb. In general, B points to an object, or part of an 
 * object, that was allocated via cublasAlloc().
 *
 * Return Values 
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if rows or cols < 0, or elemSize, lda, or 
 *                                ldb <= 0
 * HIPBLAS_STATUS_MAPPING_ERROR    if error occurred accessing GPU memory
 * HIPBLAS_STATUS_SUCCESS          if the operation completed successfully
 */
hipblasStatus_t CUBLASAPI hipblasSetMatrix (int rows, int cols, int elemSize,
                                        const void *A, int lda, void *B,
                                        int ldb)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    hipError_t cudaStat = hipSuccess;
    const char *sp = (const char *)A;
    char *dp = (char *)B;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if ((lda <= 0) || (ldb <= 0) || (elemSize <= 0) || (rows < 0) || (cols<0)){
        return HIPBLAS_STATUS_INVALID_VALUE;
    }
    /* early out if nothing to do */
    if ((rows == 0) || (cols == 0)) {
        return HIPBLAS_STATUS_SUCCESS;
    }
    if ((rows == lda) && (rows == ldb)) {
        cudaStat = hipMemcpy (dp, sp, rows * cols * elemSize, 
                               hipMemcpyHostToDevice);
    }  else {
        cudaStat = hipMemcpy2D (dp, ldb * elemSize, sp, lda * elemSize,
                                 rows * elemSize, cols,
                                 hipMemcpyHostToDevice);
    }
    if (cudaStat != hipSuccess) {
        return HIPBLAS_STATUS_MAPPING_ERROR;
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/*
 * hipblasStatus_t 
 * hipblasGetMatrix (int rows, int cols, int elemSize, const void *A, 
 *                  int lda, void *B, int ldb)
 *
 * copies a tile of rows x cols elements from a matrix A in GPU memory
 * space to a matrix B in CPU memory space. Each element requires storage
 * of elemSize bytes. Both matrices are assumed to be stored in column 
 * major format, with the leading dimension (i.e. number of rows) of 
 * source matrix A provided in lda, and the leading dimension of matrix B
 * provided in ldb. In general, A points to an object, or part of an 
 * object, that was allocated via cublasAlloc().
 *
 * Return Values 
 * -------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if rows, cols, eleSize, lda, or ldb <= 0
 * HIPBLAS_STATUS_MAPPING_ERROR    if error occurred accessing GPU memory
 * HIPBLAS_STATUS_SUCCESS          if the operation completed successfully
 */
hipblasStatus_t CUBLASAPI hipblasGetMatrix (int rows, int cols, int elemSize,
                                        const void *A, int lda, void *B,
                                        int ldb)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    hipError_t cudaStat = hipSuccess;
    const char *sp = (const char *)A;
    char *dp = (char *)B;

    if (!cublasInitialized (ctx)) {
        return HIPBLAS_STATUS_NOT_INITIALIZED;
    }
    if ((lda <= 0) || (ldb <= 0) || (elemSize <= 0) || (rows < 0) || (cols<0)){
        return HIPBLAS_STATUS_INVALID_VALUE;
    }
    /* early out if nothing to do */
    if ((rows == 0) || (cols == 0)) {
        return HIPBLAS_STATUS_SUCCESS;
    }
    if ((rows == lda) && (rows == ldb)) {
        cudaStat = hipMemcpy (dp, sp, rows * cols * elemSize, 
                               hipMemcpyDeviceToHost);
    }  else {
        cudaStat = hipMemcpy2D (dp, ldb * elemSize, sp, lda * elemSize,
                                 rows * elemSize, cols,
                                 hipMemcpyDeviceToHost);
    }
    if (cudaStat != hipSuccess) {
        return HIPBLAS_STATUS_MAPPING_ERROR;
    }
    return HIPBLAS_STATUS_SUCCESS;
}

/* -------------------------- stub functions ------------------------------- */


/* Add a GUID to the compiled library for tracking purposes */
#include "../../cuda/common/version.h"
CUDA_STAMP_GUID;

