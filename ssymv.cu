#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function ssymv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void ssymv_up_main (struct cublasSsymvParams parms);
__global__ void ssymv_lo_main (struct cublasSsymvParams parms);

/*
 * void 
 * hipblasSsymv (char uplo, int n, float alpha, const float *A, int lda, 
 *              const float *x, int incx, float beta, float *y, int incy)
 *
 * performs the matrix-vector operation
 *
 *     y = alpha*A*x + beta*y
 *
 * Alpha and beta are single precision scalars, and x and y are single 
 * precision vectors, each with n elements. A is a symmetric n x n matrix 
 * consisting of single precision elements that is stored in either upper or 
 * lower storage mode.
 *
 * Input
 * -----
 * uplo   specifies whether the upper or lower triangular part of the array A 
 *        is to be referenced. If uplo == 'U' or 'u', the symmetric matrix A 
 *        is stored in upper storage mode, i.e. only the upper triangular part
 *        of A is to be referenced while the lower triangular part of A is to 
 *        be inferred. If uplo == 'L' or 'l', the symmetric matrix A is stored
 *        in lower storage mode, i.e. only the lower triangular part of A is 
 *        to be referenced while the upper triangular part of A is to be 
 *        inferred.
 * n      specifies the number of rows and the number of columns of the 
 *        symmetric matrix A. n must be at least zero.
 * alpha  single precision scalar multiplier applied to A*x.
 * A      single precision array of dimensions (lda, n). If uplo == 'U' or 'u',
 *        the leading n x n upper triangular part of the array A must contain
 *        the upper triangular part of the symmetric matrix and the strictly
 *        lower triangular part of A is not referenced. If uplo == 'L' or 'l',
 *        the leading n x n lower triangular part of the array A must contain
 *        the lower triangular part of the symmetric matrix and the strictly
 *        upper triangular part of A is not referenced. 
 * lda    leading dimension of A. It must be at least max (1, n).
 * x      single precision array of length at least (1 + (n - 1) * abs(incx)).
 * incx   storage spacing between elements of x. incx must not be zero.
 * beta   single precision scalar multiplier applied to vector y.
 * y      single precision array of length at least (1 + (n - 1) * abs(incy)). 
 *        If beta is zero, y is not read. 
 * incy   storage spacing between elements of y. incy must not be zero.
 *
 * Output
 * ------
 * y      updated according to y = alpha*A*x + beta*y
 *
 * Reference: http://www.netlib.org/blas/ssymv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n < 0, or if incx or incy == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSsymv (char uplo, int n, float alpha,
                                     const float *A, int lda, const float *x,
                                     int incx, float beta, float *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSsymvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    } 
    else if (n < 0) {
        info = 2;
    }
    else if (lda < imax (1, n)) {
        info = 5;
    }
    else if (incx == 0) {
        info = 7;
    }
    else if (incy == 0) {
        info = 10;
    }
    if (info) {
        cublasXerbla ("SSYMV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((n == 0) || ((alpha == 0.0f) && (beta == 1.0f))) {
        return;
    }

    memset (&params, 0, sizeof(params));

    params.up = toupper(uplo) == 'U';
    params.n = n;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    params.beta = beta;
    params.y = y;
    params.incy = incy;

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.up) {
        ssymv_up_main<<<CUBLAS_SSYMV_CTAS,CUBLAS_SSYMV_THREAD_COUNT>>>(params);
    } else {
        ssymv_lo_main<<<CUBLAS_SSYMV_CTAS,CUBLAS_SSYMV_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* dimension m, counter i */
/* dimension n, counter j */

/* column-major ordering */
#define IDXA(row,col)       (parms.lda*(col)+(row))
#define IDXX(i)             (startx + ((i) * parms.incx))
#define IDXY(i)             (starty + ((i) * parms.incy))

#define X_ELEMS_PER_THREAD  (4)
#define IINC                (CUBLAS_SSYMV_CTAS * CUBLAS_SSYMV_THREAD_COUNT)
#define JINC                (CUBLAS_SSYMV_THREAD_COUNT * X_ELEMS_PER_THREAD)
#define XINC                (CUBLAS_SSYMV_THREAD_COUNT)

__shared__ float XX[JINC];  /* cached portion of vector x */

__global__ void ssymv_up_main (struct cublasSsymvParams parms) 
{
#undef  UPPER
#define UPPER 1
#include "ssymv.h"
}

__global__ void ssymv_lo_main (struct cublasSsymvParams parms)
{
#undef  UPPER
#define UPPER 0
#include "ssymv.h"
}
