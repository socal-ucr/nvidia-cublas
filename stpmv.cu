#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function stpmv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void stpmv_main_up_tr (struct cublasStpmvParams parms);
__global__ void stpmv_main_lo_tr (struct cublasStpmvParams parms);
__global__ void stpmv_main_up_nt (struct cublasStpmvParams parms);
__global__ void stpmv_main_lo_nt (struct cublasStpmvParams parms);

/*
 * void 
 * hipblasStpmv (char uplo, char trans, char diag, int n, const float *AP, 
 *              float *x, int incx);
 *
 * performs one of the matrix-vector operations x = op(A) * x, where op(A) = A,
 * or op(A) = transpose(A). x is an n element single precision vector, and A 
 * is an n x n, unit or non-unit, upper or lower triangular matrix composed 
 * of single precision elements.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix A is an upper or lower triangular
 *        matrix. If uplo == 'U' or 'u', then A is an upper triangular matrix.
 *        If uplo == 'L' or 'l', then A is a lower triangular matrix.
 * trans  specifies op(A). If transa == 'N' or 'n', op(A) = A. If trans == 'T',
 *        't', 'C', or 'c', op(A) = transpose(A)
 * diag   specifies whether or not matrix A is unit triangular. If diag == 'U'
 *        or 'u', A is assumed to be unit triangular. If diag == 'N' or 'n', A 
 *        is not assumed to be unit triangular.
 * n      specifies the number of rows and columns of the matrix A. n must be 
 *        at least zero. In the current implementation n must not exceed 4070.
 * AP     single precision array with at least ((n * (n + 1)) / 2) elements. If
 *        uplo == 'U' or 'u', the array AP contains the upper triangular part 
 *        of the symmetric matrix A, packed sequentially, column by column; 
 *        that is, if i <= j, then A[i,j] is stored in AP[i+(j*(j+1)/2)]. If 
 *        uplo == 'L' or 'L', the array AP contains the lower triangular part 
 *        of the symmetric matrix A, packed sequentially, column by column; 
 *        that is, if i >= j, then A[i,j] is stored in AP[i+((2*n-j+1)*j)/2].
 * x      single precision array of length at least (1 + (n - 1) * abs(incx)).
 *        On entry, x contains the source vector. On exit, x is overwritten 
 *        with the result vector.
 * incx   specifies the storage spacing for elements of x. incx must not be 
 *        zero.
 *
 * Output
 * ------
 * x      updated according to x = op(A) * x,
 *
 * Reference: http://www.netlib.org/blas/stpmv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx == 0, n < 0, or n > 4070
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasStpmv (char uplo, char trans, char diag, int n, 
                                     const float *AP, float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasStpmvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check other inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if ((toupper (trans) != 'N') && 
             (toupper (trans) != 'T') && 
             (toupper (trans) != 'C')) {
        info = 2;
    } 
    else if ((toupper (diag) != 'U') &&
             (toupper (diag) != 'N')) {
        info = 3;
    }
    else if ((n < 0) && (n > CUBLAS_STPMV_MAX_DIM)) {
        info = 4;
    }
    else if (incx == 0) {
        info = 7;
    }
    if (info) {
        cublasXerbla ("STPMV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if (n == 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.AP = AP;
    params.x = x;
    params.incx = incx;
    params.trans = ((toupper(trans) == 'T') || (toupper(trans) == 'C'));
    params.unit = (toupper(diag) == 'U');
    params.up = (toupper(uplo) == 'U');

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.trans) {
        if (params.up) {
            stpmv_main_up_tr<<<CUBLAS_STPMV_CTAS,CUBLAS_STPMV_THREAD_COUNT>>>(params);
        } else {
            stpmv_main_lo_tr<<<CUBLAS_STPMV_CTAS,CUBLAS_STPMV_THREAD_COUNT>>>(params);
        }
    } else {
        if (params.up) {
            stpmv_main_up_nt<<<CUBLAS_STPMV_CTAS,CUBLAS_STPMV_THREAD_COUNT>>>(params);
        } else {
            stpmv_main_lo_nt<<<CUBLAS_STPMV_CTAS,CUBLAS_STPMV_THREAD_COUNT>>>(params);
        }
    }
    cudaStat = hipGetLastError(); /* check for launch error */
    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#define IDXX(i)             (startx + ((i) * parms.incx))
#define XINC                (CUBLAS_STPMV_THREAD_COUNT)

__shared__ float xx[CUBLAS_STPMV_MAX_DIM];  /* cached portion of vector x */

__global__ void stpmv_main_up_tr (struct cublasStpmvParams parms) 
{
#undef  UP
#undef  TRANS
#define TRANS 1
#define UP    1
#include "stpmv.h"
}

__global__ void stpmv_main_lo_tr (struct cublasStpmvParams parms) 
{
#undef  UP
#undef  TRANS
#define TRANS 1
#define UP    0
#include "stpmv.h"
}

__global__ void stpmv_main_up_nt (struct cublasStpmvParams parms) 
{
#undef  UP
#undef  TRANS
#define TRANS 0
#define UP    1
#include "stpmv.h"
}

__global__ void stpmv_main_lo_nt (struct cublasStpmvParams parms) 
{
#undef  UP
#undef  TRANS
#define TRANS 0
#define UP    0
#include "stpmv.h"
}
