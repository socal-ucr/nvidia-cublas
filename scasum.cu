#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function scasum */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float2> texX;

__global__ void scasum_gld_main (struct cublasScasumParams parms);
__global__ void scasum_tex_main (struct cublasScasumParams parms);

/*
 * float 
 * hipblasScasum (int n, const cuDouble *x, int incx)
 *
 * takes the sum of the absolute values of a complex vector and returns a
 * single precision result. Note that this is not the L1 norm of the vector.
 * The result is the sum from 0 to n-1 of abs(real(x[ix+i*incx])) +
 * abs(imag(x(ix+i*incx))), where ix = 1 if incx <= 0, else ix = 1+(1-n)*incx.
 * 
 * Input
 * -----
 * n      number of elements in input vector
 * x      single-complex vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * returns the single precision sum of absolute values of real and imaginary
 * parts (0 if n <= 0 or incx <= 0, or if an error occurs)
 *
 * Reference: http://www.netlib.org/blas/scasum.f
 *
 * Error status for this function can be retrieved via cublasGetError(). 
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ float CUBLASAPI hipblasScasum (int n, const hipComplex *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasScasumParams params;
    float *devPtrT;
    hipblasStatus_t status;
    hipError_t cudaStat;
    int nbrCtas;
    int threadsPerCta;
    float sum = 0.0f;
    float *tx;
    int i;
    int useTexture;
    int sizeX = n * (imax (1, abs(incx)));
    size_t texXOfs = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return sum;
    }

    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }

    if (n < CUBLAS_SCASUM_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_SCASUM_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_SCASUM_CTAS;
         threadsPerCta = CUBLAS_SCASUM_THREAD_COUNT;
    }
    
    useTexture = sizeX < CUBLAS_MAX_1DBUF_SIZE;

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 120000) || /* experimental bound */
        ((sizeX == n) && (!(((uintptr_t) x) % CUBLAS_LONG_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) != 
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return sum;
        }
        texXOfs /= sizeof(x[0]);
    }

    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return sum;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (float *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return sum;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.incx = incx;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        scasum_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        scasum_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }

    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return sum;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
    }    

    for (i = 0; i < nbrCtas; i++) {
        sum += tx[i];
    }

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);
    return sum;   
}

__shared__ float partialSum[CUBLAS_SCASUM_THREAD_COUNT];

__global__ void scasum_gld_main (struct cublasScasumParams parms) 
{
#undef  USE_TEX
#define USE_TEX 0
#include "scasum.h"
}

__global__ void scasum_tex_main (struct cublasScasumParams parms) 
{
#undef  USE_TEX
#define USE_TEX 1
#include "scasum.h"
}
