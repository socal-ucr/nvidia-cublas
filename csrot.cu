#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function csrot */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void csrot_main (struct cublasCsrotParams parms);

/*
 * void 
 * csrot (int n, hipComplex *x, int incx, cuCumplex *y, int incy, float c, 
 *        float s)
 *
 * multiplies a 2x2 matrix ( c s) with the 2xn matrix ( transpose(x) )
 *                         (-s c)                     ( transpose(y) )
 *
 * The elements of x are in x[lx + i * incx], i = 0 ... n - 1, where lx = 1 if 
 * incx >= 0, else lx = 1 + (1 - n) * incx, and similarly for y using ly and 
 * incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single-precision complex vector with n elements
 * incx   storage spacing between elements of x
 * y      single-precision complex vector with n elements
 * incy   storage spacing between elements of y
 * c      cosine component of rotation matrix
 * s      sine component of rotation matrix
 *
 * Output
 * ------
 * x      rotated vector x (unchanged if n <= 0)
 * y      rotated vector y (unchanged if n <= 0)
 *
 * Reference  http://www.netlib.org/blas/csrot.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasCsrot (int n, hipComplex *x, int incx, 
                                     hipComplex *y, int incy, float c, float s)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCsrotParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* early out if nothing to do */
    if (n <= 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.incx = incx;
    params.cy = y;
    params.incy = incy;
    params.sc = c;
    params.ss = s;

    cublasVectorSplay (n, CUBLAS_CSROT_THREAD_MIN, CUBLAS_CSROT_THREAD_MAX,
                       CUBLAS_CSROT_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    csrot_main<<<nbrCtas,threadsPerCta>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

__global__ void csrot_main (struct cublasCsrotParams parms) 
{
    int i, n, tid, totalThreads, ctaStart;
    hipComplex w, z;
    hipComplex *cx;
    hipComplex *cy;

    /* NOTE: wrapper must ensure that parms.n > 0  */

    tid = threadIdx.x;
    n = parms.n;
    cx = parms.cx; 
    cy = parms.cy;
    totalThreads = gridDim.x*blockDim.x;
    ctaStart = blockDim.x*blockIdx.x;
   
    if ((parms.incx == 0) || (parms.incy == 0)) {
        if ((blockIdx.x == 0) && (tid == 0)) {
            if ((parms.incx == 0) && (parms.incy == 0)) {
                hipComplex tw, tz;
                w = parms.cx[0];
                z = parms.cy[0];
                for (i = 0; i < parms.n; i++) {
                    tw.x = parms.sc * w.x + parms.ss * z.x;
                    tw.y = parms.sc * w.y + parms.ss * z.y;
                    tz.x = parms.sc * z.x - parms.ss * w.x;
                    tz.y = parms.sc * z.y - parms.ss * w.y;
                    w = tw;
                    z = tz;
                }
                cx[0] = w;
                cy[0] = z;
            } else if (parms.incx == 0) {
                int ky = (parms.incy < 0) ? ((1 - parms.n) * parms.incy) : 0;
                hipComplex temp = parms.cx[0];
                hipComplex tmp2;
                for (i = 0; i < parms.n; i++) {
                    w = temp;
                    z = cy[ky];
                    temp.x = parms.sc * w.x + parms.ss * z.x;
                    temp.y = parms.sc * w.y + parms.ss * z.y;
                    tmp2.x = parms.sc * z.x - parms.ss * w.x;
                    tmp2.y = parms.sc * z.y - parms.ss * w.y;
                    cy[ky] = tmp2;
                    ky += parms.incy;
                }
                cx[0] = temp;
            } else {
                int kx = (parms.incx < 0) ? ((1 - parms.n) * parms.incx) : 0;
                hipComplex temp = parms.cy[0];
                hipComplex tmp2;
                for (i = 0; i < parms.n; i++) {
                    w = parms.cx[kx];
                    z = temp;
                    tmp2.x = parms.sc * w.x + parms.ss * z.x;
                    tmp2.y = parms.sc * w.y + parms.ss * z.y;
                    temp.x = parms.sc * z.x - parms.ss * w.x;
                    temp.y = parms.sc * z.y - parms.ss * w.y;
                    parms.cx[kx] = tmp2;
                    kx += parms.incx;
                }
                parms.cy[0] = temp;
            }
        }
    } else if ((parms.incx == parms.incy) && (parms.incx > 0)) {
        /* equal, positive, increments */
        if (parms.incx == 1) {
            hipComplex temp, tmp2;
            /* both increments equal to 1 */
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                w = cx[i];
                z = cy[i];
                temp.x = parms.sc * w.x + parms.ss * z.x;
                temp.y = parms.sc * w.y + parms.ss * z.y;
                tmp2.x = parms.sc * z.x - parms.ss * w.x;
                tmp2.y = parms.sc * z.y - parms.ss * w.y;
                cx[i] = temp;
                cy[i] = tmp2;
            }
        } else {
            /* equal, positive, non-unit increments. */
            hipComplex temp, tmp2;
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                w = cx[i*parms.incx];
                z = cy[i*parms.incx];
                temp.x = parms.sc * w.x + parms.ss * z.x;
                temp.y = parms.sc * w.y + parms.ss * z.y;
                tmp2.x = parms.sc * z.x - parms.ss * w.x;
                tmp2.y = parms.sc * z.y - parms.ss * w.y;
                cx[i*parms.incx] = temp;
                cy[i*parms.incy] = tmp2;
            }
        }
    } else {
        /* unequal or nonpositive increments */
        hipComplex temp, tmp2;
        int ix = ((parms.incx < 0) ? ((1 - n) * parms.incx) : 0);
        int iy = ((parms.incy < 0) ? ((1 - n) * parms.incy) : 0);
        for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
            w = cx[ix+i*parms.incx];
            z = cy[iy+i*parms.incy];
            temp.x = parms.sc * w.x + parms.ss * z.x;
            temp.y = parms.sc * w.y + parms.ss * z.y;
            tmp2.x = parms.sc * z.x - parms.ss * w.x;
            tmp2.y = parms.sc * z.y - parms.ss * w.y;
            cx[ix+i*parms.incx] = temp;
            cy[iy+i*parms.incy] = tmp2;
        }
    }
}

