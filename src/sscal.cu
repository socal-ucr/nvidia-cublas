#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function sscal */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;

__global__ void sscal_gld_main (struct cublasSscalParams parms);
__global__ void sscal_tex_main (struct cublasSscalParams parms);

/*
 * void
 * hipblasSscal (int n, float alpha, float *x, int incx)
 *
 * replaces single precision vector x with single precision alpha * x. For i 
 * = 0 to n - 1, it replaces x[ix + i * incx] with alpha * x[ix + i * incx], 
 * where ix = 1 if incx >= 0, else ix = 1 + (1 - n) * incx.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * alpha  single precision scalar multiplier
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * x      single precision result (unchanged if n <= 0 or incx <= 0)
 *
 * Reference: http://www.netlib.org/blas/sscal.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 * 
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSscal (int n, float alpha, float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSscalParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;
    int sizeX = n * (imax (1, abs(incx)));
    size_t texXOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }
    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return;
    }
    
    useTexture = sizeX < CUBLAS_MAX_1DBUF_SIZE;

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 100000) || /* experimental bound */
        ((sizeX == n) && (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) != 
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texXOfs /= sizeof(x[0]);
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.sa = alpha;
    params.incx = incx;
    params.texXOfs = (int)texXOfs;

    cublasVectorSplay (n, CUBLAS_SSCAL_THREAD_MIN, CUBLAS_SSCAL_THREAD_MAX,
                       CUBLAS_SSCAL_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        sscal_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        sscal_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */
    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}

__global__ void sscal_gld_main (struct cublasSscalParams parms)
{
#undef  USE_TEX
#define USE_TEX 0
#include "sscal.h"
}

__global__ void sscal_tex_main (struct cublasSscalParams parms)
{
#undef  USE_TEX
#define USE_TEX 1
#include "sscal.h"
}
