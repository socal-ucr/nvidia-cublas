#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function crot */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void crot_main (struct cublasCrotParams parms);

/*
 * hipblasCrot (int n, hipComplex *x, int incx, hipComplex *y, int incy, float sc,
 *             hipComplex cs)
 *
 * multiplies a 2x2 matrix ( sc       cs) with the 2xn matrix ( transpose(x) )
 *                         (-conj(cs) sc)                     ( transpose(y) )
 *
 * The elements of x are in x[lx + i * incx], i = 0 ... n - 1, where lx = 1 if 
 * incx >= 0, else lx = 1 + (1 - n) * incx, and similarly for y using ly and 
 * incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single-precision complex vector with n elements
 * incx   storage spacing between elements of x
 * y      single-precision complex vector with n elements
 * incy   storage spacing between elements of y
 * sc     single-precision cosine component of rotation matrix
 * cs     single-precision complex sine component of rotation matrix
 *
 * Output
 * ------
 * x      rotated single-precision complex vector x (unchanged if n <= 0)
 * y      rotated single-precision complex vector y (unchanged if n <= 0)
 *
 * Reference: http://netlib.org/lapack/explore-html/crot.f.html
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasCrot (int n, hipComplex *x, int incx, 
                                    hipComplex *y, int incy, float sc, 
                                    hipComplex cs)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCrotParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* early out if nothing to do */
    if (n <= 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.incx = incx;
    params.cy = y;
    params.incy = incy;
    params.sc = sc;
    params.cs = cs;

    cublasVectorSplay (n, CUBLAS_CROT_THREAD_MIN, CUBLAS_CROT_THREAD_MAX,
                       CUBLAS_CROT_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    crot_main<<<nbrCtas,threadsPerCta>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

__global__ void crot_main (struct cublasCrotParams parms) 
{
    int i, n, tid, totalThreads, ctaStart;
    hipComplex w, z, conjugCs;
    hipComplex *cx;
    hipComplex *cy;
    hipComplex cc;

    /* NOTE: wrapper must ensure that parms.n > 0  */

    tid = threadIdx.x;
    n = parms.n;
    cx = parms.cx; 
    cy = parms.cy;
    totalThreads = gridDim.x * blockDim.x;
    ctaStart = blockDim.x * blockIdx.x;
    cc = make_hipComplex (parms.sc, 0.0f);
    conjugCs = hipConjf(parms.cs);
   
    if ((parms.incx == 0) || (parms.incy == 0)) {
        if ((blockIdx.x == 0) && (tid == 0)) {
            if ((parms.incx == 0) && (parms.incy == 0)) {
                hipComplex tw, tz;
                w = cx[0];
                z = cy[0];
                for (i = 0; i < parms.n; i++) {
                    tw = hipCaddf (hipCmulf(cc,w), hipCmulf (parms.cs, z));
                    tz = hipCsubf (hipCmulf(cc,z), hipCmulf (conjugCs, w));
                    w = tw;
                    z = tz;
                }
                cx[0] = w;
                cy[0] = z;
            } else if (parms.incx == 0) {
                int ky = (parms.incy < 0) ? ((1 - parms.n) * parms.incy) : 0;
                hipComplex temp = cx[0];
                for (i = 0; i < parms.n; i++) {
                    w = temp;
                    z = cy[ky];
                    temp  = hipCaddf (hipCmulf (cc, w), hipCmulf (parms.cs, z));
                    cy[ky]= hipCsubf (hipCmulf (cc, z), hipCmulf (conjugCs, w));
                    ky += parms.incy;
                }
                cx[0] = temp;
            } else {
                int kx = (parms.incx < 0) ? ((1 - parms.n) * parms.incx) : 0;
                hipComplex temp = cy[0];
                for (i = 0; i < parms.n; i++) {
                    w = cx[kx];
                    z = temp;
                    cx[kx] = hipCaddf (hipCmulf (cc, w), hipCmulf (parms.cs, z));
                    temp   = hipCsubf (hipCmulf (cc, z), hipCmulf (conjugCs, w));
                    kx += parms.incx;
                }
                cy[0] = temp;
            }
        }
    } else if ((parms.incx == parms.incy) && (parms.incx > 0)) {
        /* equal, positive, increments */
        if (parms.incx == 1) {
            /* both increments equal to 1 */
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                w = cx[i];
                z = cy[i];
                cx[i] = hipCaddf (hipCmulf (cc, w), hipCmulf (parms.cs, z));
                cy[i] = hipCsubf (hipCmulf (cc, z), hipCmulf (conjugCs, w));
            }
        } else {
            /* equal, positive, non-unit increments. */
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                w = cx[i*parms.incx];
                z = cy[i*parms.incx];
                cx[i*parms.incx]=hipCaddf (hipCmulf(cc,w), hipCmulf(parms.cs, z));
                cy[i*parms.incx]=hipCsubf (hipCmulf(cc,z), hipCmulf(conjugCs, w));
            }
        }
    } else {
        /* unequal or nonpositive increments */
        int ix = ((parms.incx < 0) ? ((1 - n) * parms.incx) : 0);
        int iy = ((parms.incy < 0) ? ((1 - n) * parms.incy) : 0);
        for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
            w = cx[ix+i*parms.incx];
            z = cy[iy+i*parms.incy];
            cx[ix+i*parms.incx] = hipCaddf (hipCmulf(cc,w), hipCmulf(parms.cs,z));
            cy[iy+i*parms.incy] = hipCsubf (hipCmulf(cc,z), hipCmulf(conjugCs,w));
        }
    }
}

