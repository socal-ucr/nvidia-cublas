#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function sdot */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;
texture<float> texY;

__global__ void sdot_gld_main (struct cublasSdotParams parms);
__global__ void sdot_tex_main (struct cublasSdotParams parms);

/*
 * float 
 * sdot (int n, const float *x, int incx, const float *y, int incy)
 *
 * computes the dot product of two single precision vectors. It returns the 
 * dot product of the single precision vectors x and y if successful, and
 * 0.0f otherwise. It computes the sum for i = 0 to n - 1 of x[lx + i * 
 * incx] * y[ly + i * incy], where lx = 1 if incx >= 0, else lx = 1 + (1 - n)
 * *incx, and ly is defined in a similar way using incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 * y      single precision vector with n elements
 * incy   storage spacing between elements of y
 *
 * Output
 * ------
 * returns single precision dot product (zero if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/sdot.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has nor been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to execute on GPU
 */
__host__ float CUBLASAPI hipblasSdot (int n, const float *x, int incx,
                                     const float *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSdotParams params;
    hipError_t cudaStat;
    hipblasStatus_t status;
    float *devPtrT;
    int nbrCtas;
    int threadsPerCta;
    float *tx;
    float dot = 0.0f;
    int i;
    int sizeX = n * (imax (1, abs(incx)));
    int sizeY = n * (imax (1, abs(incy)));
    size_t texXOfs = 0;
    size_t texYOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return dot;
    }
    
    if (n < CUBLAS_SDOT_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_SDOT_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_SDOT_CTAS;
         threadsPerCta = CUBLAS_SDOT_THREAD_COUNT;
    }

    /* early out if nothing to do */
    if (n <= 0) {
        return dot;
    }

    useTexture = ((sizeX < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeY < CUBLAS_MAX_1DBUF_SIZE));

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 70000) || /* experimental bound */
        ((sizeX == n) && (sizeY == n) && 
         (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)) && 
         (!(((uintptr_t) y) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        if ((cudaStat=hipBindTexture (&texYOfs,texY,y,sizeY*sizeof(y[0]))) !=
            hipSuccess) {
            hipUnbindTexture (texX);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        texXOfs /= sizeof(x[0]);
        texYOfs /= sizeof(y[0]);
    }

    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return dot;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (float *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return dot;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.sx = x;
    params.incx = incx;
    params.sy = y;
    params.incy = incy;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;
    params.texYOfs = (int)texYOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        sdot_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        sdot_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return dot;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        return dot;
    }    

    for (i = 0; i < nbrCtas; i++) {
        dot += tx[i];
    }

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texY)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }

    return dot;
}

__shared__ float partialSum[CUBLAS_SDOT_THREAD_COUNT];        

__global__ void sdot_gld_main (struct cublasSdotParams parms) 
{
#undef  USE_TEX
#define USE_TEX 0
#include "sdot.h"
}

__global__ void sdot_tex_main (struct cublasSdotParams parms) 
{
#undef  USE_TEX
#define USE_TEX 1
#include "sdot.h"
}
