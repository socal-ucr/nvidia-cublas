#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function ddot */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "cublas_v1.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<double> texX;
texture<double> texY;

__global__ void ddot_gld_main (struct cublasDdotParams parms);
__global__ void ddot_tex_main (struct cublasDdotParams parms);

/*
 * double 
 * ddot (int n, const double *x, int incx, const double *y, int incy)
 *
 * computes the dot product of two single precision vectors. It returns the 
 * dot product of the single precision vectors x and y if successful, and
 * 0.0f otherwise. It computes the sum for i = 0 to n - 1 of x[lx + i * 
 * incx] * y[ly + i * incy], where lx = 1 if incx >= 0, else lx = 1 + (1 - n)
 * *incx, and ly is defined in a similar way using incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 * y      single precision vector with n elements
 * incy   storage spacing between elements of y
 *
 * Output
 * ------
 * returns single precision dot product (zero if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/ddot.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has nor been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to execute on GPU
 */
__host__ double CUBLASAPI hipblasDdot (int n, const double *x, int incx,
                                     const double *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasDdotParams params;
    hipError_t cudaStat;
    hipblasStatus_t status;
    double *devPtrT;
    int nbrCtas;
    int threadsPerCta;
    double *tx;
    double dot = 0.0f;
    int i;
    int sizeX = n * (imax (1, abs(incx)));
    int sizeY = n * (imax (1, abs(incy)));
    size_t texXOfs = 0;
    size_t texYOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return dot;
    }
    
    if (n < CUBLAS_DDOT_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_DDOT_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_DDOT_CTAS;
         threadsPerCta = CUBLAS_DDOT_THREAD_COUNT;
    }

    /* early out if nothing to do */
    if (n <= 0) {
        return dot;
    }

    useTexture = ((sizeX < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeY < CUBLAS_MAX_1DBUF_SIZE));

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 70000) || /* experimental bound */
        ((sizeX == n) && (sizeY == n) && 
         (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)) && 
         (!(((uintptr_t) y) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        if ((cudaStat=hipBindTexture (&texYOfs,texY,y,sizeY*sizeof(y[0]))) !=
            hipSuccess) {
            hipUnbindTexture (texX);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        texXOfs /= sizeof(x[0]);
        texYOfs /= sizeof(y[0]);
    }

    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return dot;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (double *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return dot;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.sx = x;
    params.incx = incx;
    params.sy = y;
    params.incy = incy;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;
    params.texYOfs = (int)texYOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        ddot_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        ddot_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return dot;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        return dot;
    }    

    for (i = 0; i < nbrCtas; i++) {
        dot += tx[i];
    }

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texY)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }

    return dot;
}

__shared__ double partialSum[CUBLAS_DDOT_THREAD_COUNT];        

__global__ void ddot_gld_main (struct cublasDdotParams parms) 
{
#undef  USE_TEX
#define USE_TEX 0
#include "ddot.h"
}

__global__ void ddot_tex_main (struct cublasDdotParams parms) 
{
#undef  USE_TEX
#define USE_TEX 1
#include "ddot.h"
}
