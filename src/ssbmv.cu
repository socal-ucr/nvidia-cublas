#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function ssbmv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void ssbmvu_main (struct cublasSsbmvParams parms);
__global__ void ssbmvl_main (struct cublasSsbmvParams parms);

/*
 * void 
 * hipblasSsbmv (char uplo, int n, int k, float alpha, const float *A, int lda,
 *              const float *x, int incx, float beta, float *y, int incy)
 *
 * performs the matrix-vector operation
 *
 *     y := alpha*A*x + beta*y
 *
 * alpha and beta are single precision scalars. x and y are single precision
 * vectors with n elements. A is an n by n symmetric band matrix consisting 
 * of single precision elements, with k super-diagonals and the same number
 * of subdiagonals.
 *
 * Input
 * -----
 * uplo   specifies whether the upper or lower triangular part of the symmetric
 *        band matrix A is being supplied. If uplo == 'U' or 'u', the upper 
 *        triangular part is being supplied. If uplo == 'L' or 'l', the lower 
 *        triangular part is being supplied.
 * n      specifies the number of rows and the number of columns of the
 *        symmetric matrix A. n must be at least zero.
 * k      specifies the number of super-diagonals of matrix A. Since the matrix
 *        is symmetric, this is also the number of sub-diagonals. k must be at
 *        least zero.
 * alpha  single precision scalar multiplier applied to A*x.
 * A      single precision array of dimensions (lda, n). When uplo == 'U' or 
 *        'u', the leading (k + 1) x n part of array A must contain the upper
 *        triangular band of the symmetric matrix, supplied column by column,
 *        with the leading diagonal of the matrix in row (k+1) of the array,
 *        the first super-diagonal starting at position 2 in row k, and so on.
 *        The top left k x k triangle of the array A is not referenced. When
 *        uplo == 'L' or 'l', the leading (k + 1) x n part of the array A must
 *        contain the lower triangular band part of the symmetric matrix, 
 *        supplied column by column, with the leading diagonal of the matrix in
 *        row 1 of the array, the first sub-diagonal starting at position 1 in
 *        row 2, and so on. The bottom right k x k triangle of the array A is
 *        not referenced.
 * lda    leading dimension of A. lda must be at least (k + 1).
 * x      single precision array of length at least (1 + (n - 1) * abs(incx)).
 * incx   storage spacing between elements of x. incx must not be zero.
 * beta   single precision scalar multiplier applied to vector y. If beta is 
 *        zero, y is not read.
 * y      single precision array of length at least (1 + (n - 1) * abs(incy)). 
 *        If beta is zero, y is not read.
 * incy   storage spacing between elements of y. incy must not be zero.
 *
 * Output
 * ------
 * y      updated according to alpha*A*x + beta*y
 *
 * Reference: http://www.netlib.org/blas/ssbmv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_INVALID_VALUE    if k or n < 0, or if incx or incy == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSsbmv (char uplo, int n, int k, float alpha, 
                                     const float *A, int lda, const float *x, 
                                     int incx, float beta, float *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSsbmvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    } 
    else if (n < 0) {
        info = 2;
    }
    else if (k < 0) {
        info = 3;
    }
    else if (lda < (k + 1)) {
        info = 6;
    }
    else if (incx == 0) {
        info = 8;
    }
    else if (incy == 0) {
        info = 11;
    }
    if (info) {
        cublasXerbla ("SSBMV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((n == 0) || ((alpha == 0.0f) && (beta == 1.0f))) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.up = toupper(uplo) == 'U';
    params.n = n;
    params.k = k;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    params.beta = beta;
    params.y = y;
    params.incy = incy;

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.up) {
        ssbmvu_main<<<CUBLAS_SSBMV_CTAS,CUBLAS_SSBMV_THREAD_COUNT>>>(params);
    } else {
        ssbmvl_main<<<CUBLAS_SSBMV_CTAS,CUBLAS_SSBMV_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

#define IDXA_UP(row,col)    ((parms.lda*(col))+(parms.k)+((row)-(col)))
#define IDXA_LO(row,col)    ((parms.lda*(col))+((row)-(col)))
#define IDXX(i)             (startx + ((i) * parms.incx))
#define IDXY(i)             (starty + ((i) * parms.incy))

#define X_ELEMS_PER_THREAD  (4)
#define IINC                (CUBLAS_SSBMV_CTAS * CUBLAS_SSBMV_THREAD_COUNT)
#define JINC                (CUBLAS_SSBMV_THREAD_COUNT * X_ELEMS_PER_THREAD)
#define XINC                (CUBLAS_SSBMV_THREAD_COUNT)

__shared__ float XX[JINC];  /* cached portion of vector x */

__global__ void ssbmvu_main (struct cublasSsbmvParams parms) 
{
    int i, ii, j, jj, idx, incr, tid;
    float sdot;
    int startx;
    int starty;

    /*
     * NOTE: wrapper must ensure that parms.n >= 0, and that parms.incx and 
     *       parms.incy are != 0 
     */

    tid = threadIdx.x;
    startx = (parms.incx >= 0) ? 0 : ((1 - parms.n) * parms.incx);
    starty = (parms.incy >= 0) ? 0 : ((1 - parms.n) * parms.incy);
    for (i = 0; i < parms.n; i += IINC) {
        /* first row being processed by this CTA */
        ii = i + blockIdx.x * CUBLAS_SSBMV_THREAD_COUNT;
        if (ii >= parms.n) break; /* nothing to do for this CTA */
        ii += tid; /* row being processed by this thread */
        sdot = 0.0f; /* initialize dot product handled by this thread */
        /* iterate over chunks of rows. These chunks are very large, so
         * in many case we'll only executed the loop body once, i.e. we'll
         * process the whole row in one fell swoop.
         */
        for (j = 0; j < parms.n; j += JINC) {
            int jjLimit = min (j + JINC, parms.n);
            incr = XINC * parms.incx;
            jj = j + tid;

            __syncthreads ();
            idx = IDXX(jj);
#if (X_ELEMS_PER_THREAD == 4)
            if (jj < (jjLimit - 3 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
                XX[tid+3*XINC] = parms.alpha * parms.x[idx + 3 * incr];
            }
            else if (jj < (jjLimit - 2 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
            }
            else if (jj < (jjLimit - 1 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
            }
            else if (jj < (jjLimit - 0 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
            }
#else
#error current code cannot handle X_ELEMS_PER_THREAD != 4
#endif
            __syncthreads ();
            
            if (ii < parms.n) { /* if this row is active, accumulate dp */
                int jjStart = ii - parms.k; /* may be negative */
                int jjEnd   = ii + parms.k + 1;
                if ((jjEnd > j) && (jjStart < jjLimit)) {
                    jj = max (j, jjStart);
                    while (jj < (min (jjLimit, jjEnd))) {
                        int idx = (ii < jj) ? IDXA_UP(ii,jj) : IDXA_UP(jj,ii);
                        sdot += parms.A[idx] * XX[jj-j];
                        jj++;
                    }
                }
            }
        }
        if (ii < parms.n) { /* if this row is active, write out dp */
            idx = IDXY(ii);
            if (parms.beta != 0.0f) {
                sdot += parms.beta * parms.y[idx];
            }
            parms.y[idx] = sdot;
        }
    }
}

__global__ void ssbmvl_main (struct cublasSsbmvParams parms) 
{
    int i, ii, j, jj, idx, incr, tid;
    float sdot;
    int startx;
    int starty;

    /*
     * NOTE: wrapper must ensure that parms.n >= 0, and that parms.incx and 
     *       parms.incy are != 0 
     */

    tid = threadIdx.x;
    startx = (parms.incx >= 0) ? 0 : ((1 - parms.n) * parms.incx);
    starty = (parms.incy >= 0) ? 0 : ((1 - parms.n) * parms.incy);
    for (i = 0; i < parms.n; i += IINC) {
        /* first row being processed by this CTA */
        ii = i + blockIdx.x * CUBLAS_SSBMV_THREAD_COUNT;
        if (ii >= parms.n) break; /* nothing to do for this CTA */
        ii += tid; /* row being processed by this thread */
        sdot = 0.0f; /* initialize dot product handled by this thread */
        /* iterate over chunks of rows. These chunks are very large, so
         * in many case we'll only executed the loop body once, i.e. we'll
         * process the whole row in one fell swoop.
         */
        for (j = 0; j < parms.n; j += JINC) {
            int jjLimit = min (j + JINC, parms.n);
            incr = XINC * parms.incx;
            jj = j + tid;
            __syncthreads ();
            idx = IDXX(jj);
#if (X_ELEMS_PER_THREAD == 4)
            if (jj < (jjLimit - 3 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
                XX[tid+3*XINC] = parms.alpha * parms.x[idx + 3 * incr];
            }
            else if (jj < (jjLimit - 2 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
            }
            else if (jj < (jjLimit - 1 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
            }
            else if (jj < (jjLimit - 0 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
            }
#else
#error current code cannot handle X_ELEMS_PER_THREAD != 4
#endif
            __syncthreads ();
            
            if (ii < parms.n) { /* if this row is active, accumulate dp */
                int jjStart = ii - parms.k; /* may be negative */
                int jjEnd   = ii + parms.k + 1;
                if ((jjEnd > j) && (jjStart < jjLimit)) {
                    jj = max (j, jjStart);
                    while (jj < (min (jjLimit, jjEnd))) {
                        int idx = (ii > jj) ? IDXA_LO(ii,jj) : IDXA_LO(jj,ii);
                        sdot += parms.A[idx] * XX[jj-j];
                        jj++;
                    }
                }
            }
        }
        if (ii < parms.n) { /* if this row is active, write out dp */
            idx = IDXY(ii);
            if (parms.beta != 0.0f) {
                sdot += parms.beta * parms.y[idx];
            }
            parms.y[idx] = sdot;
        }
    }
}


