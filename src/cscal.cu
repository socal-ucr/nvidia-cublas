#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function cscal */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void cscal_main (struct cublasCscalParams parms);

/*
 * void
 * hipblasCscal (int n, hipComplex alpha, hipComplex *x, int incx)
 *
 * replaces single-complex vector x with single-complex alpha * x. For i 
 * = 0 to n - 1, it replaces x[ix + i * incx] with alpha * x[ix + i * incx], 
 * where ix = 1 if incx >= 0, else ix = 1 + (1 - n) * incx.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * alpha  single-complex scalar multiplier
 * x      single-complex vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * x      single-complex result (unchanged if n <= 0 or incx <= 0)
 *
 * Reference: http://www.netlib.org/blas/cscal.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 * 
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasCscal (int n, hipComplex alpha, hipComplex *x,
                                     int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCscalParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }
    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.ca = alpha;
    params.incx = incx;

    cublasVectorSplay (n, CUBLAS_CSCAL_THREAD_MIN, CUBLAS_CSCAL_THREAD_MAX,
                       CUBLAS_CSCAL_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    cscal_main<<<nbrCtas,threadsPerCta>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

__global__ void cscal_main (struct cublasCscalParams parms)
{
    int i, n, tid, totalThreads, ctaStart;
    hipComplex *cx;

    /* NOTE: wrapper must ensure that parms.n > 0 and parms.incx > 0 */
    tid = threadIdx.x;
    n = parms.n;
    cx = parms.cx;
    totalThreads = gridDim.x * blockDim.x;
    ctaStart = blockDim.x * blockIdx.x;

    if (parms.incx == 1) {
        /* increment equal to 1 */
        for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
            cx[i] = hipCmulf (parms.ca, cx[i]);
        }
    } else {
        /* increment not equal to 1 */
        int ix = ((parms.incx < 0) ? ((1 - n) * parms.incx) : 0);
        for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
            cx[ix+i*parms.incx] = hipCmulf (parms.ca, cx[ix+i*parms.incx]);
        }
    }
}
