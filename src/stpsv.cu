#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function stpsv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void stpsv_main_up_tr (struct cublasStpsvParams parms);
__global__ void stpsv_main_lo_tr (struct cublasStpsvParams parms);
__global__ void stpsv_main_up_nt (struct cublasStpsvParams parms);
__global__ void stpsv_main_lo_nt (struct cublasStpsvParams parms);

/*
 * void 
 * hipblasStpsv (char uplo, char trans, char diag, int n, const float *AP, 
 *              float *X, int incx)
 *
 * solves one of the systems of equations op(A)*x = b, where op(A) is either 
 * op(A) = A or op(A) = transpose(A). b and x are n element vectors, and A is
 * an n x n unit or non-unit, upper or lower triangular matrix. No test for
 * singularity or near-singularity is included in this routine. Such tests 
 * must be performed before calling this routine.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix is an upper or lower triangular matrix
 *        as follows: If uplo == 'U' or 'u', A is an upper triangluar matrix.
 *        If uplo == 'L' or 'l', A is a lower triangular matrix.
 * trans  specifies op(A). If trans == 'N' or 'n', op(A) = A. If trans == 'T',
 *        't', 'C', or 'c', op(A) = transpose(A).
 * diag   specifies whether A is unit triangular. If diag == 'U' or 'u', A is
 *        assumed to be unit triangular; thas is, diagonal elements are not
 *        read and are assumed to be unity. If diag == 'N' or 'n', A is not
 *        assumed to be unit triangular.
 * n      specifies the number of rows and columns of the matrix A. n must be
 *        at least zero.
 * AP     single precision array with at least ((n*(n+1))/2) elements. If uplo
 *        == 'U' or 'u', the array AP contains the upper triangular matrix A,
 *        packed sequentially, column by column; that is, if i <= j, then 
 *        A[i,j] is stored is AP[i+(j*(j+1)/2)]. If uplo == 'L' or 'L', the 
 *        array AP contains the lower triangular matrix A, packed sequentially,
 *        column by column; that is, if i >= j, then A[i,j] is stored in 
 *        AP[i+((2*n-j+1)*j)/2]. When diag = 'U' or 'u', the diagonal elements
 *        of A are not referenced and are assumed to be unity.
 * x      single precision array of length at least (1+(n-1)*abs(incx)).
 * incx   storage spacing between elements of x. It must not be zero.
 *
 * Output
 * ------
 * x      updated to contain the solution vector x that solves op(A) * x = b.
 *
 * Reference: http://www.netlib.org/blas/stpsv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx == 0 or if n < 0 or n > 4070
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasStpsv (char uplo, char trans, char diag, int n,
                                     const float *AP, float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasStpsvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check other inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if ((toupper (trans) != 'N') && 
             (toupper (trans) != 'T') && 
             (toupper (trans) != 'C')) {
        info = 2;
    } 
    else if ((toupper (diag) != 'U') &&
             (toupper (diag) != 'N')) {
        info = 3;
    }
    /* FIXME: There should be no upper limit on n */
    else if ((n < 0) || (n > CUBLAS_STPSV_MAX_DIM)) {
        info = 4;
    }
    else if (incx == 0) {
        info = 7;
    }
    if (info) {
        cublasXerbla ("STPSV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if (n == 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.AP = AP;
    params.x = x;
    params.incx = incx;
    params.trans = ((toupper(trans) == 'T') || (toupper(trans) == 'C'));
    params.unit = (toupper(diag) == 'U');
    params.up = (toupper(uplo) == 'U');

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.up) {
        if (params.trans) {
            stpsv_main_up_tr<<<CUBLAS_STPSV_CTAS,CUBLAS_STPSV_THREAD_COUNT>>>(params);
        } else {
            stpsv_main_up_nt<<<CUBLAS_STPSV_CTAS,CUBLAS_STPSV_THREAD_COUNT>>>(params);
        }
    } else {
         if (params.trans) {
            stpsv_main_lo_tr<<<CUBLAS_STPSV_CTAS,CUBLAS_STPSV_THREAD_COUNT>>>(params);
        } else {
            stpsv_main_lo_nt<<<CUBLAS_STPSV_CTAS,CUBLAS_STPSV_THREAD_COUNT>>>(params);
        }
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#define IDXX(i)             (startx + ((i) * parms.incx))
#define XINC                (CUBLAS_STPSV_THREAD_COUNT)

__shared__ float XX[CUBLAS_STPSV_MAX_DIM];  /* cached portion of vector x */
__shared__ float temp;                      /* current diagonal element */


__global__ void stpsv_main_up_tr (struct cublasStpsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    1
#include "stpsv.h"                
}

__global__ void stpsv_main_lo_tr (struct cublasStpsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    0
#include "stpsv.h"                
}

__global__ void stpsv_main_up_nt (struct cublasStpsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    1
#include "stpsv.h"                
}

__global__ void stpsv_main_lo_nt (struct cublasStpsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    0
#include "stpsv.h"                
}


