#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function sspr */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void sspr_up_main (struct cublasSsprParams parms);
__global__ void sspr_lo_main (struct cublasSsprParams parms);

/*
 * void 
 * hipblasSspr (char uplo, int n, float alpha, const float *x, int incx, 
 *             float *AP)
 *
 * performs the symmetric rank 1 operation
 * 
 *    A = alpha * x * transpose(x) + A,
 * 
 * where alpha is a single precision scalar and x is an n element single 
 * precision vector. A is a symmetric n x n matrix consisting of single 
 * precision elements that is supplied in packed form.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix data is stored in the upper or the lower
 *        triangular part of array AP. If uplo == 'U' or 'u', then the upper 
 *        triangular part of A is supplied in AP. If uplo == 'L' or 'l', then 
 *        the lower triangular part of A is supplied in AP.
 * n      specifies the number of rows and columns of the matrix A. It must be
 *        at least zero.
 * alpha  single precision scalar multiplier applied to x * transpose(x).
 * x      single precision array of length at least (1 + (n - 1) * abs(incx)).
 * incx   storage spacing between elements of x. incx must not be zero.
 * AP     single precision array with at least ((n * (n + 1)) / 2) elements. If
 *        uplo == 'U' or 'u', the array AP contains the upper triangular part 
 *        of the symmetric matrix A, packed sequentially, column by column; 
 *        that is, if i <= j, then A[i,j] is stored is AP[i+(j*(j+1)/2)]. If 
 *        uplo == 'L' or 'L', the array AP contains the lower triangular part 
 *        of the symmetric matrix A, packed sequentially, column by column; 
 *        that is, if i >= j, then A[i,j] is stored in AP[i+((2*n-j+1)*j)/2].
 *
 * Output
 * ------
 * A      updated according to A = alpha * x * transpose(x) + A
 *
 * Reference: http://www.netlib.org/blas/sspr.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n < 0, or incx == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSspr (char uplo, int n, float alpha,
                                    const float *x, int incx, float *AP)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSsprParams params;
    hipError_t cudaStat;
    int up;
    int info = 0;
    dim3 ctaDims(CUBLAS_SSPR_GRIDW, CUBLAS_SSPR_GRIDH);

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    up = toupper(uplo) == 'U';

    info = 0;
    if ((!up) && (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if (n < 0) {
        info = 2;
    }
    else if (incx == 0) {
        info = 5;
    }
    if (info) {
        cublasXerbla ("SSPR  ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((n == 0) || (alpha == 0.0f)) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.up = up;
    params.n = n;
    params.alpha = alpha;
    params.AP = AP;
    params.x = x;
    params.incx = incx;
    
    cudaStat = hipGetLastError(); /* clear error status */
    if (params.up) {
        sspr_up_main<<<ctaDims,CUBLAS_SSPR_THREAD_COUNT>>>(params);
    } else {
        sspr_lo_main<<<ctaDims,CUBLAS_SSPR_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#undef IDXX
#define IDXX(i)             (startx + ((i) * parms.incx))

#define BLK_LOG             (5)
#define BLK                 (1 << BLK_LOG)
#define ELEMS_PER_THREAD    ((BLK*BLK)/CUBLAS_SSPR_THREAD_COUNT)
#define IIINC               (BLK)
#define JJINC               (BLK)
#define IINC                (IIINC*CUBLAS_SSPR_GRIDH)
#define JINC                (JJINC*CUBLAS_SSPR_GRIDW)
#define A_NBR_COLS          (CUBLAS_SSPR_THREAD_COUNT/IIINC)

#if (BLK & (BLK - 1))
#error tile dimension must be a power of two
#endif

#if (CUBLAS_SSPR_THREAD_COUNT < BLK)
#error thread count must be greater than or equal to tile dimension
#endif

#if ((BLK*BLK)%CUBLAS_SSPR_THREAD_COUNT)
#error number of tile elements must be integral multiple of thread count
#endif

#if (CUBLAS_SSPR_THREAD_COUNT%IIINC)
#error thread count must be integral multple of tile dimension
#endif

__shared__ float xi[IINC];
__shared__ float xj[IINC];

__global__ void sspr_up_main (struct cublasSsprParams parms) 
{
#undef LOWER
#define LOWER 0
#include "sspr.h"
}

__global__ void sspr_lo_main (struct cublasSsprParams parms)
{
#undef LOWER
#define LOWER 1
#include "sspr.h"
}
