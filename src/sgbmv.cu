#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function sgbmv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void sgbmvn_main (struct cublasSgbmvParams parms);
__global__ void sgbmvt_main (struct cublasSgbmvParams parms);

/*
 * void 
 * hipblasSgbmv (char trans, int m, int n, int kl, int ku, float alpha,
 *              const float *A, int lda, const float *x, int incx, float beta,
 *              float *y, int incy);
 *
 * performs one of the matrix-vector operations
 *
 *    y = alpha*op(A)*x + beta*y,  op(A)=A or op(A) = transpose(A)
 *
 * alpha and beta are single precision scalars. x and y are single precision
 * vectors. A is an m by n band matrix consisting of single precision elements
 * with kl sub-diagonals and ku super-diagonals.
 *
 * Input
 * -----
 * trans  specifies op(A). If trans == 'N' or 'n', op(A) = A. If trans == 'T', 
 *        't', 'C', or 'c', op(A) = transpose(A)
 * m      specifies the number of rows of the matrix A. m must be at least 
 *        zero.
 * n      specifies the number of columns of the matrix A. n must be at least
 *        zero.
 * kl     specifies the number of sub-diagonals of matrix A. It must be at 
 *        least zero.
 * ku     specifies the number of super-diagonals of matrix A. It must be at
 *        least zero.
 * alpha  single precision scalar multiplier applied to op(A).
 * A      single precision array of dimensions (lda, n). The leading
 *        (kl + ku + 1) x n part of the array A must contain the band matrix A,
 *        supplied column by column, with the leading diagonal of the matrix 
 *        in row (ku + 1) of the array, the first super-diagonal starting at 
 *        position 2 in row ku, the first sub-diagonal starting at position 1
 *        in row (ku + 2), and so on. Elements in the array A that do not 
 *        correspond to elements in the band matrix (such as the top left 
 *        ku x ku triangle) are not referenced.
 * lda    leading dimension of A. lda must be at least (kl + ku + 1).
 * x      single precision array of length at least (1+(n-1)*abs(incx)) when 
 *        trans == 'N' or 'n' and at least (1+(m-1)*abs(incx)) otherwise.
 * incx   specifies the increment for the elements of x. incx must not be zero.
 * beta   single precision scalar multiplier applied to vector y. If beta is 
 *        zero, y is not read.
 * y      single precision array of length at least (1+(m-1)*abs(incy)) when 
 *        trans == 'N' or 'n' and at least (1+(n-1)*abs(incy)) otherwise. If 
 *        beta is zero, y is not read.
 * incy   On entry, incy specifies the increment for the elements of y. incy 
 *        must not be zero.
 *
 * Output
 * ------
 * y      updated according to y = alpha*op(A)*x + beta*y
 *
 * Reference: http://www.netlib.org/blas/sgbmv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n < 0, or if incx or incy == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSgbmv (char trans, int m, int n, int kl, int ku, 
                                     float alpha, const float *A, int lda, 
                                     const float *x, int incx, float beta, 
                                     float *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSgbmvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check inputs */
    if ((toupper (trans) != 'N') &&
        (toupper (trans) != 'T') &&
        (toupper (trans) != 'C')) {
        info = 1;
    } 
    else if (m < 0) {
        info = 2;
    }
    else if (n < 0) {
        info = 3;
    }
    else if (kl < 0) {
        info = 4;
    }
    else if (ku < 0) {
        info = 5;
    }
    else if (lda < (kl + ku + 1)) {
        info = 8;
    }
    else if (incx == 0) {
        info = 10;
    }
    else if (incy == 0) {
        info = 13;
    }
    if (info) {
        cublasXerbla ("SGBMV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((m == 0) || (n == 0) || ((alpha == 0.0f) && (beta == 1.0f))) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.trans = toupper(trans) != 'N';
    params.m = m;
    params.n = n;
    params.kl = kl;
    params.ku = ku;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    params.beta = beta;
    params.y = y;
    params.incy = incy;

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.trans) {
        sgbmvt_main<<<CUBLAS_SGBMV_CTAS,CUBLAS_SGBMV_THREAD_COUNT>>>(params);
    } else {
        sgbmvn_main<<<CUBLAS_SGBMV_CTAS,CUBLAS_SGBMV_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

#define IDXA(row,col)       ((parms.lda*(col))+((row)-(col)+(parms.ku)))
#define IDXX(i)             (startx + ((i) * parms.incx))
#define IDXY(i)             (starty + ((i) * parms.incy))

#define X_ELEMS_PER_THREAD  (4)
#define IINC                (CUBLAS_SGBMV_CTAS * CUBLAS_SGBMV_THREAD_COUNT)
#define JINC                (CUBLAS_SGBMV_THREAD_COUNT * X_ELEMS_PER_THREAD)
#define XINC                (CUBLAS_SGBMV_THREAD_COUNT)

__shared__ float XX[JINC];  /* cached portion of vector x */

__global__ void sgbmvn_main (struct cublasSgbmvParams parms) 
{
    int i, ii, j, jj, idx, incr, tid;
    float sdot;
    int startx;
    int starty;

    /*
     * NOTE: wrapper must ensure that parms.n >= 0, and that parms.incx and 
     *       parms.incy are != 0 
     */

    tid = threadIdx.x;
    startx = (parms.incx >= 0) ? 0 : ((1 - parms.n) * parms.incx);
    starty = (parms.incy >= 0) ? 0 : ((1 - parms.m) * parms.incy);
    for (i = 0; i < parms.m; i += IINC) {
        /* first row being processed by this CTA */
        ii = i + blockIdx.x * CUBLAS_SGBMV_THREAD_COUNT;
        if (ii >= parms.m) break; /* nothing to do for this CTA */
        ii += tid; /* row being processed by this thread */
        sdot = 0.0f; /* initialize dot product handled by this thread */
        /* iterate over chunks of rows. These chunks are very large, so
         * in many case we'll only executed the loop body once, i.e. we'll
         * process the whole row in one fell swoop.
         */
        for (j = 0; j < parms.n; j += JINC) {
            int jjLimit = min (j + JINC, parms.n);
            incr = XINC * parms.incx;
            jj = j + tid;
            __syncthreads ();
            idx = IDXX(jj);
#if (X_ELEMS_PER_THREAD == 4)
            if (jj < (jjLimit - 3 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
                XX[tid+3*XINC] = parms.alpha * parms.x[idx + 3 * incr];
            }
            else if (jj < (jjLimit - 2 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
            }
            else if (jj < (jjLimit - 1 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
            }
            else if (jj < (jjLimit - 0 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
            }
#else
#error current code cannot handle X_ELEMS_PER_THREAD != 4
#endif
            __syncthreads ();
            
            if (ii < parms.m) { /* if this row is active, accumulate dp */
                int jjStart = ii - parms.kl; /* may be negative */
                int jjEnd   = ii + parms.ku + 1;
                if ((jjEnd > j) && (jjStart < jjLimit)) {
                    jj = max (j, jjStart);
//                        printf ("row %d initial j=%d less than %d\n", ii, jj, (min (jjLimit, jjEnd)));fflush(stdout);
                    while (jj < (min (jjLimit, jjEnd))) {
//                            printf ("%d,%d A[%d]=%g\n",ii,jj,IDXA(ii,jj),parms.A[IDXA(ii,jj)]);fflush(stdout);
                        sdot += parms.A[IDXA(ii,jj)] * XX[jj-j];
                        jj++;
                    }
                }
            }
        }
        if (ii < parms.m) { /* if this row is active, write out dp */
            idx = IDXY(ii);
            if (parms.beta != 0.0f) {
                sdot += parms.beta * parms.y[idx];
            }
            parms.y[idx] = sdot;
        }
    }
}

__global__ void sgbmvt_main (struct cublasSgbmvParams parms) 
{
    int i, ii, j, jj, idx, incr, tid;
    float sdot;
    int startx;
    int starty;

    /*
     * NOTE: wrapper must ensure that parms.n >= 0, and that parms.incx and 
     *       parms.incy are != 0 
     */

    tid = threadIdx.x;
    startx = (parms.incx >= 0) ? 0 : ((1 - parms.m) * parms.incx);
    starty = (parms.incy >= 0) ? 0 : ((1 - parms.n) * parms.incy);
    for (i = 0; i < parms.n; i += IINC) {
        /* first row being processed by this CTA */
        ii = i + blockIdx.x * CUBLAS_SGBMV_THREAD_COUNT;
        if (ii >= parms.n) break; /* nothing to do for this CTA */
        ii += tid; /* row being processed by this thread */
        sdot = 0.0f; /* initialize dot product handled by this thread */
        /* iterate over chunks of rows. These chunks are very large, so
         * in many case we'll only executed the loop body once, i.e. we'll
         * process the whole row in one fell swoop.
         */
        for (j = 0; j < parms.m; j += JINC) {
            int jjLimit = min (j + JINC, parms.m);
            incr = XINC * parms.incx;
            jj = j + tid;
            __syncthreads ();
            idx = IDXX(jj);
#if (X_ELEMS_PER_THREAD == 4)
            if (jj < (jjLimit - 3 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
                XX[tid+3*XINC] = parms.alpha * parms.x[idx + 3 * incr];
            }
            else if (jj < (jjLimit - 2 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
                XX[tid+2*XINC] = parms.alpha * parms.x[idx + 2 * incr];
            }
            else if (jj < (jjLimit - 1 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
                XX[tid+1*XINC] = parms.alpha * parms.x[idx + 1 * incr];
            }
            else if (jj < (jjLimit - 0 * XINC)) {
                XX[tid+0*XINC] = parms.alpha * parms.x[idx + 0 * incr];
//                printf ("XX[%d]=%g\n",tid+0*XINC,XX[tid+0*XINC]);fflush(stdout);
            }
#else
#error current code cannot handle X_ELEMS_PER_THREAD != 4
#endif
            __syncthreads ();
            
            if (ii < parms.n) { /* if this row is active, accumulate dp */
                int jjStart = ii - parms.ku; /* may be negative */
                int jjEnd   = ii + parms.kl + 1;
                if ((jjEnd > j) && (jjStart < jjLimit)) {
                    jj = max (j, jjStart);
//                    printf ("col %d initial j=%d less than %d\n", ii, jj, (min (jjLimit, jjEnd)));fflush(stdout);
                    while (jj < (min (jjLimit, jjEnd))) {
//                        printf ("%d,%d A[%d]=%g\n",jj,ii,IDXA(jj,ii),parms.A[IDXA(jj,ii)]);fflush(stdout);
                        sdot += parms.A[IDXA(jj,ii)] * XX[jj-j];
                        jj++;
                    }
                }
            }
        }
        if (ii < parms.n) { /* if this row is active, write out dp */
            idx = IDXY(ii);
            if (parms.beta != 0.0f) {
                sdot += parms.beta * parms.y[idx];
            }
            parms.y[idx] = sdot;
        }
    }
}


