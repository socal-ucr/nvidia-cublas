#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function ssyr */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

#if (CUBLAS_SSYR_GRIDW!=CUBLAS_SSYR_GRIDH)
#error super tile is not square!
#endif

#define TILE_DIM_LOG        (5)
#define TILE_DIM            (1 << (TILE_DIM_LOG))
#define TILE_SIZE           (TILE_DIM*TILE_DIM)
#define SUP_TILE_DIM        (TILE_DIM*CUBLAS_SSYR_GRIDW)

#if ((TILE_SIZE%CUBLAS_SSYR_THREAD_COUNT)!=0)
#error TILE_SIZE and CUBLAS_SSYR_THREAD_COUNT do not divide evenly!
#endif
#if ((CUBLAS_SSYR_THREAD_COUNT%TILE_DIM)!=0)
#error CUBLAS_SSYR_THREAD_COUNT and TILE_DIM do not divide evenly!
#endif

#define COL_INCR            (CUBLAS_SSYR_THREAD_COUNT/TILE_DIM)
#define ELEMS_PER_THREAD    (TILE_SIZE/CUBLAS_SSYR_THREAD_COUNT)

__global__ void ssyr_main_up_sw (struct cublasSsyrParams parms);
__global__ void ssyr_main_lo_sw (struct cublasSsyrParams parms);
__global__ void ssyr_main_up_hw (struct cublasSsyrParams parms);
__global__ void ssyr_main_lo_hw (struct cublasSsyrParams parms);

/*
 * void 
 * hipblasSsyr (char uplo, int n, float alpha, const float *x, int incx, 
 *             float *A, int lda)
 *
 * performs the symmetric rank 1 operation
 *
 *    A = alpha * x * transpose(x) + A,
 *
 * where alpha is a single precision scalar, x is an n element single 
 * precision vector and A is an n x n symmetric matrix consisting of 
 * single precision elements. Matrix A is stored in column major format,
 * and lda is the leading dimension of the two-dimensional array 
 * containing A.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix data is stored in the upper or 
 *        the lower triangular part of array A. If uplo = 'U' or 'u',
 *        then only the upper triangular part of A may be referenced.
 *        If uplo = 'L' or 'l', then only the lower triangular part of
 *        A may be referenced.
 * n      specifies the number of rows and columns of the matrix A. It
 *        must be at least 0.
 * alpha  single precision scalar multiplier applied to x * transpose(x)
 * x      single precision array of length at least (1 + (n - 1) * abs(incx))
 * incx   specifies the storage spacing between elements of x. incx must 
 *        not be zero.
 * A      single precision array of dimensions (lda, n). If uplo = 'U' or 
 *        'u', then A must contain the upper triangular part of a symmetric 
 *        matrix, and the strictly lower triangular part is not referenced. 
 *        If uplo = 'L' or 'l', then A contains the lower triangular part 
 *        of a symmetric matrix, and the strictly upper triangular part is 
 *        not referenced.
 * lda    leading dimension of the two-dimensional array containing A. lda
 *        must be at least max(1, n).
 *
 * Output
 * ------
 * A      updated according to A = alpha * x * transpose(x) + A
 *
 * Reference: http://www.netlib.org/blas/ssyr.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n < 0, or incx == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSsyr (char uplo, int n, float alpha,
                                    const float *x, int incx, float *A, 
                                    int lda)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSsyrParams params;
    hipError_t cudaStat;
    int up;
    int info = 0;
    int usePureHwStepper = 0;
    dim3 ctaDimsHw (((n+TILE_DIM-1)/TILE_DIM), ((n+TILE_DIM-1)/TILE_DIM));
    dim3 ctaDimsSw (CUBLAS_SSYR_GRIDW, CUBLAS_SSYR_GRIDH);

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    info = 0;
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if (n < 0) {
        info = 2;
    }
    else if (incx == 0) {
        info = 5;
    }
    else if (lda < imax (1, n)) {
        info = 7;
    }
    if (info) {
        cublasXerbla ("SSYR  ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((n == 0) || (alpha == 0.0f)) {
        return;
    }

    /* choose HW-only stepping if dimensions of result matrix do not exceed the
     * maximum CTA grid dimensions.
     */
    usePureHwStepper = n < (CUBLAS_CTA_MAX_DIM * TILE_DIM);

    /* HW grid never seems to be a win on G80, maybe because the supertile has 
       better spatial locality ?
    */
    usePureHwStepper = 0; 

    memset (&params, 0, sizeof(params));
    up = toupper(uplo) == 'U';

    params.up = up;
    params.n = n;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    
    cudaStat = hipGetLastError(); /* clear error status */
    if (usePureHwStepper) {
        if (params.up) {
            ssyr_main_up_hw<<<ctaDimsHw,CUBLAS_SSYR_THREAD_COUNT>>>(params);
        } else {
            ssyr_main_lo_hw<<<ctaDimsHw,CUBLAS_SSYR_THREAD_COUNT>>>(params);
        }
    } else {
        if (params.up) {
            ssyr_main_up_sw<<<ctaDimsSw,CUBLAS_SSYR_THREAD_COUNT>>>(params);
        } else {
            ssyr_main_lo_sw<<<ctaDimsSw,CUBLAS_SSYR_THREAD_COUNT>>>(params);
        }
    }       
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#undef IDXA
#undef IDXX
#define IDXA(row,col) (parms.lda*(col)+(row))
#define IDXX(i)       (startx+((i)*parms.incx))

__shared__ float xi[TILE_DIM];
__shared__ float xj[TILE_DIM];

__global__ void ssyr_main_up_hw (struct cublasSsyrParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  LOWER
#define LOWER              0
#define USE_MIXED_STEPPER  0
#include "ssyr.h"
}

__global__ void ssyr_main_lo_hw (struct cublasSsyrParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  LOWER
#define LOWER              1
#define USE_MIXED_STEPPER  0
#include "ssyr.h"
}

__global__ void ssyr_main_up_sw (struct cublasSsyrParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  LOWER
#define LOWER              0
#define USE_MIXED_STEPPER  1
#include "ssyr.h"
}

__global__ void ssyr_main_lo_sw (struct cublasSsyrParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  LOWER
#define LOWER              1
#define USE_MIXED_STEPPER  1
#include "ssyr.h"
}
