#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function sasum */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;

__global__ void sasum_gld_main (struct cublasSasumParams parms);
__global__ void sasum_tex_main (struct cublasSasumParams parms);

/*
 * float 
 * sasum (int n, const float *x, int incx)
 *
 * computes the sum of the absolute values of the elements of single 
 * precision vector x; that is, the result is the sum from i = 0 to n - 1 of 
 * abs(x[1 + i * incx]).
 * 
 * Input
 * -----
 * n      number of elements in input vector
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * returns the single precision sum of absolute values
 * (0 if n <= 0 or incx <= 0, or if an error occurs)
 *
 * Reference: http://www.netlib.org/blas/sasum.f
 *
 * Error status for this function can be retrieved via cublasGetError(). 
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ float CUBLASAPI hipblasSasum (int n, const float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSasumParams params;
    float *devPtrT;
    hipblasStatus_t status;
    hipError_t cudaStat;
    int nbrCtas;
    int threadsPerCta;
    int sizeX = n * (imax (1, abs(incx)));
    size_t texXOfs = 0;
    float sum = 0.0f;
    float *tx;
    int i;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return sum;
    }

    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }

    if (n < CUBLAS_SASUM_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_SASUM_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_SASUM_CTAS;
         threadsPerCta = CUBLAS_SASUM_THREAD_COUNT;
    }

    useTexture = sizeX < CUBLAS_MAX_1DBUF_SIZE;

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 140000) || /* experimental bound */
        ((sizeX == n) && (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) != 
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return sum;
        }
        texXOfs /= sizeof(x[0]);
    }
    
    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return sum;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (float *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return sum;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.incx = incx;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        sasum_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        sasum_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return sum;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
    }    

    for (i = 0; i < nbrCtas; i++) {
        sum += tx[i];
    }

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);
    return sum;   
}

__shared__ float partialSum[CUBLAS_SASUM_THREAD_COUNT];

__global__ void sasum_gld_main (struct cublasSasumParams parms) 
{
#undef  USE_TEX
#define USE_TEX 0
#include "sasum.h"
}

__global__ void sasum_tex_main (struct cublasSasumParams parms) 
{
#undef  USE_TEX
#define USE_TEX 1
#include "sasum.h"
}
