#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function snrm2 */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

#define CUBLAS_SNRM2_STATE_ZERO    0
#define CUBLAS_SNRM2_STATE_TINY    1
#define CUBLAS_SNRM2_STATE_NORMAL  2
#define CUBLAS_SNRM2_STATE_HUGE    3
#define CUBLAS_SNRM2_STATE_DONE    4

texture<float> texX;

__global__ void snrm2_gld_main (struct cublasSnrm2Params parms);
__global__ void snrm2_tex_main (struct cublasSnrm2Params parms);
__host__ static float local_snrm2 (int n, const float *sx, int incx);

/*
 * float 
 * snrm2 (int n, const float *x, int incx)
 *
 * computes the Euclidean norm of the single precision n-vector x (with
 * storage increment incx). This code uses a multiphase model of 
 * accumulation to avoid intermediate underflow and overflow.
 *
 * Input
 * -----
 * n      number of elements in input vector
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * returns Euclidian norm (0 if n <= 0 or incx <= 0, or if an error occurs)
 *
 * Reference: http://www.netlib.org/blas/snrm2.f
 * Reference: http://www.netlib.org/slatec/lin/snrm2.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 * 
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ float CUBLASAPI hipblasSnrm2 (int n, const float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSnrm2Params params;
    float *devPtrT;
    hipblasStatus_t status;
    hipError_t cudaStat;
    int nbrCtas;
    int threadsPerCta;
    float sum = 0.0f;
    float *tx;
    int sizeX = n * (imax (1, abs(incx)));
    size_t texXOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return sum;
    }

    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }

    if (n < CUBLAS_SNRM2_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_SNRM2_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_SNRM2_CTAS;
         threadsPerCta = CUBLAS_SNRM2_THREAD_COUNT;
    }

    useTexture = sizeX < CUBLAS_MAX_1DBUF_SIZE;

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 130000) || /* experimental bound */
        ((sizeX == n) && (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return sum;
        }
        texXOfs /= sizeof(x[0]);
    }
    
    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return sum;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (float *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return sum;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.incx = incx;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        snrm2_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        snrm2_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return sum;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        return sum;
    }    
    
    sum = local_snrm2 (nbrCtas, tx, 1);

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }

    return sum;   
}

/*
 * float snrm2 (int n, const float *sx, int incx);
 *
 * IN:  n    number of elements in input vector
 *      sx   single precision vector with n elements
 *      incx storage spacing between elements of sx
 *
 * OUT: returns Euclidian norm (0 if n <= 0 or incx <= 0)
 *
 * BLAS level 1, see http://www.netlib.org/blas/snrm2.f
 *
 * Euclidean norm of the n-vector stored in sx with storage
 * increment incx. This version uses a multi-phase model of
 * accumulation to avoid intermediate underflow and overflow.
 * See http://www.netlib.org/slatec/lin/snrm2.f
 */
__host__ static float local_snrm2 (int n, const float *sx, int incx)
{
    float cutLo = 4.441e-16f;
    float cutHi = 1.304e+19f;
    unsigned int i, ns, state;
    volatile float sum = 0.0f;
    volatile float hiTest;
    volatile float t = 0.0f;
    volatile float ta = 0.0f;
    volatile float xmax = 0.0f;
    volatile float xmaxRecip;

    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }
    ns = n * incx;
    hiTest = cutHi / (float)n;
    i = 0;
    state = CUBLAS_SNRM2_STATE_ZERO;
    while (state != CUBLAS_SNRM2_STATE_DONE) {
        switch (state) {
        case CUBLAS_SNRM2_STATE_ZERO:
            while ((i < ns) && ((t = sx[i]) == 0.0f)) {
                i += incx;
            }
            if (i >= ns) {
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {
                state = CUBLAS_SNRM2_STATE_TINY;
            }
            break;            
        case CUBLAS_SNRM2_STATE_TINY:
            xmax = (float)fabs(t);
            xmaxRecip = 1.0f / xmax;
            while ((i < ns) && ((ta = (float)fabs(t = sx[i])) < cutLo)) {
                if (ta > xmax) {
                    /* Adjust scale factor */
                    t = xmax / t;
                    sum = 1.0f + sum * t * t;
                    xmax = ta;
                    xmaxRecip = 1.0f / xmax;
                } else {
                    t = t * xmaxRecip;
                    sum += t * t;
                }
                i += incx; 
            }
            if (i >= ns) {
                sum = (float)sqrt(sum);
                sum = xmax * sum;
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {            
                state = CUBLAS_SNRM2_STATE_NORMAL;
            }
            break;
        case CUBLAS_SNRM2_STATE_NORMAL:
            sum = (sum * xmax) * xmax;
            while ((i < ns) && ((ta = (float)fabs(t = sx[i])) < hiTest)) {
                sum += t * t;
                i += incx;  
            }
            if (i >= ns) {
                sum = (float)sqrt(sum);
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {
                state = CUBLAS_SNRM2_STATE_HUGE;
            }
            break;
        case CUBLAS_SNRM2_STATE_HUGE:
            xmax = ta;
            xmaxRecip = 1.0f / xmax;
            sum = (sum * xmaxRecip) * xmaxRecip;
            while (i < ns) {
                t = sx[i];
                ta = (float)fabs(t);
                if (ta > xmax) {
                    /* Adjust scale factor */
                    t = xmax / t;
                    sum = 1.0f + sum * t * t;
                    xmax = ta;
                    xmaxRecip = 1.0f / xmax;
                } else {
                    t = t * xmaxRecip;
                    sum += t * t;
                }
                i += incx; 
            }
            sum = (float)sqrt (sum);
            sum = xmax * sum;
            state = CUBLAS_SNRM2_STATE_DONE;
            break;
        }
    }
    return sum;
}

__shared__ float partialSum[CUBLAS_SNRM2_THREAD_COUNT];

__global__ void snrm2_gld_main (struct cublasSnrm2Params parms) 
{
#undef USE_TEX
#define USE_TEX 0
#include "snrm2.h"
}

__global__ void snrm2_tex_main (struct cublasSnrm2Params parms) 
{
#undef USE_TEX
#define USE_TEX 1
#include "snrm2.h"
}
