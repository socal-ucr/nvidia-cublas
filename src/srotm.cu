#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function srotm */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;
texture<float> texY;

__global__ void srotm_gld_main (struct cublasSrotmParams parms);
__global__ void srotm_tex_main (struct cublasSrotmParams parms);

/*
 * void 
 * hipblasSrotm (int n, float *x, int incx, float *y, int incy, 
 *              const float* sparam)
 *
 * applies the modified Givens transformation, h, to the 2 x n matrix
 *
 *    ( transpose(x) )
 *    ( transpose(y) )
 *
 * The elements of x are in x[lx + i * incx], i = 0 to n-1, where lx = 1 if 
 * incx >= 0, else lx = 1 + (1 - n) * incx, and similarly for y using ly and 
 * incy. With sparam[0] = sflag, h has one of the following forms:
 *
 *        sflag = -1.0f   sflag = 0.0f    sflag = 1.0f    sflag = -2.0f
 *
 *        (sh00  sh01)    (1.0f  sh01)    (sh00  1.0f)    (1.0f  0.0f)
 *    h = (          )    (          )    (          )    (          )
 *        (sh10  sh11)    (sh10  1.0f)    (-1.0f sh11)    (0.0f  1.0f)
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 * y      single precision vector with n elements
 * incy   storage spacing between elements of y
 * sparam 5-element vector. sparam[0] is sflag described above. sparam[1] 
 *        through sparam[4] contain the 2x2 rotation matrix h: sparam[1]
 *        contains sh00, sparam[2] contains sh10, sparam[3] contains sh01,
 *        and sprams[4] contains sh11.
 *
 * Output
 * ------
 * x     rotated vector x (unchanged if n <= 0)
 * y     rotated vector y (unchanged if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/srotm.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSrotm (int n, float *x, int incx, float *y,
                                     int incy, const float* sparam)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSrotmParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;
    int sizeX = n * (imax (1, abs(incx)));
    int sizeY = n * (imax (1, abs(incy)));
    size_t texXOfs = 0;
    size_t texYOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }
    /* early out if nothing to do */
    if (n <= 0) {
        return;
    }

    useTexture = ((sizeX < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeY < CUBLAS_MAX_1DBUF_SIZE));

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 50000) || /* experimental bound */
        ((sizeX == n) && (sizeY == n) && 
         (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)) && 
         (!(((uintptr_t) y) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        if ((cudaStat=hipBindTexture (&texYOfs,texY,y,sizeY*sizeof(y[0]))) !=
            hipSuccess) {
            hipUnbindTexture (texX);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texXOfs /= sizeof(x[0]);
        texYOfs /= sizeof(y[0]);
    }
 
    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.incx = incx;
    params.sy = y;
    params.incy = incy;
    params.texXOfs = (int)texXOfs;
    params.texYOfs = (int)texYOfs;
    memcpy (params.sparams, sparam, sizeof(params.sparams));

    cublasVectorSplay (n, CUBLAS_SROTM_THREAD_MIN, CUBLAS_SROTM_THREAD_MAX,
                       CUBLAS_SROTM_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        srotm_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        srotm_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
       
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texY)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}

__global__ void srotm_gld_main (struct cublasSrotmParams parms)
{
#undef  USE_TEX
#define USE_TEX 0
#include "srotm.h"
}

__global__ void srotm_tex_main (struct cublasSrotmParams parms)
{
#undef  USE_TEX
#define USE_TEX 1
#include "srotm.h"
}

