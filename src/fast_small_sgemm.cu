#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-3 function sgemm */

/* Use square 16x16 tiles to access and cache portions of source matrices A,B 
 * and result matrix C
 */
#include <stdio.h>
#include <ctype.h>
#include <string.h>
#include "cublasP.h"
#define TILE_DIM_LOG    (4)
#define THREAD_COUNT    (CUBLAS_SGEMM_SMALL_THREAD_COUNT)
#include "sgemm_sizing.h"
#include "sgemm_common.h"

texture<float> texA;
texture<float> texB;

__shared__ float AA[(TILE_DIM+1)*TILE_DIM]; /*pad to elim. GRF bank conflicts*/
__shared__ float BB[(TILE_DIM+1)*TILE_DIM]; /*pad to elim. GRF bank conflicts*/

typedef void (*pf) (struct cublasSgemmParams parms);

static pf sgemm[16] = {
    fast_small_sgemm_gld_main_hw_na_nb,
    fast_small_sgemm_gld_main_hw_na_tb,
    fast_small_sgemm_gld_main_hw_ta_nb,
    fast_small_sgemm_gld_main_hw_ta_tb,
    fast_small_sgemm_gld_main_hw_na_nb_fulltile,
    fast_small_sgemm_gld_main_hw_na_tb_fulltile,
    fast_small_sgemm_gld_main_hw_ta_nb_fulltile,
    fast_small_sgemm_gld_main_hw_ta_tb_fulltile,
    fast_small_sgemm_tex_main_hw_na_nb,
    fast_small_sgemm_tex_main_hw_na_tb,
    fast_small_sgemm_tex_main_hw_ta_nb,
    fast_small_sgemm_tex_main_hw_ta_tb,
    fast_small_sgemm_tex_main_hw_na_nb_fulltile,
    fast_small_sgemm_tex_main_hw_na_tb_fulltile,
    fast_small_sgemm_tex_main_hw_ta_nb_fulltile,
    fast_small_sgemm_tex_main_hw_ta_tb_fulltile,
};             

__host__ void cublasSmallSgemm (struct cublasContext *ctx, char transa, 
                                char transb, int m, int n, int k, 
                                float alpha, const float *A, int lda, 
                                const float *B, int ldb, float beta, float *C, 
                                int ldc)
{
    struct cublasSgemmParams params;
    hipError_t cudaStat;
    int fullTilesOnly;
    size_t texAOfs = 0;
    size_t texBOfs = 0;
    int sizeA = lda * ((toupper(transa) == 'N') ? k : m);
    int sizeB = ldb * ((toupper(transb) == 'N') ? n : k);
    int useTexture;
    int funcIdx;
    dim3 ctaDimsHw (((n+TILE_DIM-1)/TILE_DIM), ((m+TILE_DIM-1)/TILE_DIM));

    useTexture = ((sizeA < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeB < CUBLAS_MAX_1DBUF_SIZE));

    /* currently, texture binding is too expensive, so using texture fetches
     * is a net negative for the small cases handled here.
     */
    useTexture = 0;

    if (useTexture){
        if ((cudaStat=hipBindTexture (&texAOfs,texA,A,sizeA*sizeof(A[0]))) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        if ((cudaStat=hipBindTexture (&texBOfs,texB,B,sizeB*sizeof(B[0]))) != hipSuccess) {
            hipUnbindTexture (texA);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texAOfs /= sizeof(A[0]);
        texBOfs /= sizeof(B[0]);
    }

    memset (&params, 0, sizeof(params));
    params.transa = ((toupper(transa) == 'T') || (toupper(transa) == 'C'));
    params.transb = ((toupper(transb) == 'T') || (toupper(transb) == 'C'));
    params.m = m;
    params.n = n;
    params.k = k;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.B = B;
    params.ldb = ldb;
    params.beta = beta;
    params.C = C;
    params.ldc =ldc;
    params.texAOfs = (int)texAOfs;
    params.texBOfs = (int)texBOfs;

    /* we can eliminate checking for endcases if we know all tiles are fully
     * populated. Important benchmark case!
     */
    fullTilesOnly = (((params.m % TILE_DIM) == 0) &&
                     ((params.n % TILE_DIM) == 0) &&
                     ((params.k % TILE_DIM) == 0));

    funcIdx = ((useTexture << 3) | (fullTilesOnly << 2) | 
               (params.transa << 1) | params.transb);

    cudaStat = hipGetLastError(); /* clear error status */
    sgemm[funcIdx]<<<ctaDimsHw,THREAD_COUNT>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texA)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texB)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}
    
__global__ void fast_small_sgemm_tex_main_hw_na_nb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_na_tb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_ta_nb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_ta_tb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_na_nb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_na_tb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_ta_nb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}

__global__ void fast_small_sgemm_tex_main_hw_ta_tb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           1
#include "sgemm.h"
}
    
__global__ void fast_small_sgemm_gld_main_hw_na_nb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_na_tb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_ta_nb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_ta_tb (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_na_nb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_na_tb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_ta_nb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}

__global__ void fast_small_sgemm_gld_main_hw_ta_tb_fulltile (struct cublasSgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         1
#define USE_TEX           0
#include "sgemm.h"
}
