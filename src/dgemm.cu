#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Use square 32x32 tiles to access and cache portions of source matrices A,B 
 * and result matrix C
 */
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include "cublasP.h"
#define TILE_DIM_LOG    (5)
#define THREAD_COUNT    (CUBLAS_DGEMM_LARGE_THREAD_COUNT)
#include "dgemm_sizing.h"
#include "dgemm_common.h"

texture<float> texA;
texture<float> texB;

__shared__ float AA[(TILE_DIM+1)*TILE_DIM]; /*pad to elim. GRF bank conflicts*/
__shared__ float BB[(TILE_DIM+1)*TILE_DIM]; /*pad to elim. GRF bank conflicts*/

typedef void (*pf) (struct cublasDgemmParams parms);
static pf dgemm_sw[16] = {
    dgemm_main_gld_sw_na_nb,
    dgemm_main_gld_sw_na_tb,
    dgemm_main_gld_sw_ta_nb,
    dgemm_main_gld_sw_ta_tb,
    dgemm_main_gld_sw_na_nb_fulltile,
    dgemm_main_gld_sw_na_tb_fulltile,
    dgemm_main_gld_sw_ta_nb_fulltile,
    dgemm_main_gld_sw_ta_tb_fulltile,
    dgemm_main_tex_sw_na_nb,
    dgemm_main_tex_sw_na_tb,
    dgemm_main_tex_sw_ta_nb,
    dgemm_main_tex_sw_ta_tb,
    dgemm_main_tex_sw_na_nb_fulltile,
    dgemm_main_tex_sw_na_tb_fulltile,
    dgemm_main_tex_sw_ta_nb_fulltile,
    dgemm_main_tex_sw_ta_tb_fulltile
};

static pf dgemm_hw[16] = {
    dgemm_main_gld_hw_na_nb,
    dgemm_main_gld_hw_na_tb,
    dgemm_main_gld_hw_ta_nb,
    dgemm_main_gld_hw_ta_tb,
    dgemm_main_gld_hw_na_nb_fulltile,
    dgemm_main_gld_hw_na_tb_fulltile,
    dgemm_main_gld_hw_ta_nb_fulltile,
    dgemm_main_gld_hw_ta_tb_fulltile,
    dgemm_main_tex_hw_na_nb,
    dgemm_main_tex_hw_na_tb,
    dgemm_main_tex_hw_ta_nb,
    dgemm_main_tex_hw_ta_tb,
    dgemm_main_tex_hw_na_nb_fulltile,
    dgemm_main_tex_hw_na_tb_fulltile,
    dgemm_main_tex_hw_ta_nb_fulltile,
    dgemm_main_tex_hw_ta_tb_fulltile
};

__host__ void cublasLargeDgemm (struct cublasContext *ctx, char transa, 
                                char transb, int m, int n, int k, 
                                float alpha, const float *A, int lda, 
                                const float *B, int ldb, float beta, float *C, 
                                int ldc)
{
    struct cublasDgemmParams params;
    hipError_t cudaStat;
    int fullTilesOnly;
    size_t texAOfs = 0;
    size_t texBOfs = 0;
    int sizeA = lda * ((toupper(transa) == 'N') ? k : m);
    int sizeB = ldb * ((toupper(transb) == 'N') ? n : k);
    int useTexture;
    int usePureHwStepper;
    int funcIdx;
    dim3 ctaDimsHw (((n+TILE_DIM-1)/TILE_DIM), ((m+TILE_DIM-1)/TILE_DIM));
    dim3 ctaDimsSw (CUBLAS_DGEMM_GRIDW, CUBLAS_DGEMM_GRIDH);

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* We can only use texture if the matrices fit into the largest matrix 
     * size supported.
     */
    useTexture = ((sizeA < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeB < CUBLAS_MAX_1DBUF_SIZE));

    /* choose HW-only stepping if dimensions of result matrix do not exceed the
     * maximum CTA grid dimensions.
     */
    usePureHwStepper = ((m < (CUBLAS_CTA_MAX_DIM * TILE_DIM)) &&
                        (n < (CUBLAS_CTA_MAX_DIM * TILE_DIM)));

    /* we can eliminate checking for endcases if we know all tiles are fully
     * populated. Important benchmark case!
     */
    fullTilesOnly = (((m % TILE_DIM) == 0) &&
                     ((n % TILE_DIM) == 0) &&
                     ((k % TILE_DIM) == 0));


    /* For matrices where each row is aligned, GLD coalesces nicely and is
     * faster, so don't use texture.
     */
    if (!(((ptrdiff_t)A)%CUBLAS_WORD_ALIGN) && 
        !(((ptrdiff_t)B)%CUBLAS_WORD_ALIGN) &&
        !(lda%(CUBLAS_WORD_ALIGN/sizeof(A[0]))) &&
        !(ldb%(CUBLAS_WORD_ALIGN/sizeof(B[0])))) {
        useTexture = 0;
    }

    if (useTexture){
        if ((cudaStat=hipBindTexture (&texAOfs,texA,A,sizeA*sizeof(A[0]))) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        if ((cudaStat=hipBindTexture (&texBOfs,texB,B,sizeB*sizeof(B[0]))) != hipSuccess) {
            hipUnbindTexture (texA);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texAOfs /= sizeof(A[0]);
        texBOfs /= sizeof(B[0]);
    }

    memset (&params, 0, sizeof(params));
    params.transa = ((toupper(transa) == 'T') || (toupper(transa) == 'C'));
    params.transb = ((toupper(transb) == 'T') || (toupper(transb) == 'C'));
    params.m = m;
    params.n = n;
    params.k = k;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.B = B;
    params.ldb = ldb;
    params.beta = beta;
    params.C = C;
    params.ldc =ldc;
    params.texAOfs = (int)texAOfs;
    params.texBOfs = (int)texBOfs;

    funcIdx = ((useTexture << 3) | (fullTilesOnly << 2) | 
               (params.transa << 1) | params.transb);

    cudaStat = hipGetLastError(); /* clear error status */
    if (usePureHwStepper) {
        dgemm_hw[funcIdx]<<<ctaDimsHw,THREAD_COUNT>>>(params);
    } else {
        dgemm_sw[funcIdx]<<<ctaDimsSw,THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texA)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texB)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}

__global__ void dgemm_main_gld_hw_na_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_na_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_ta_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_ta_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY   
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_na_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_na_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_ta_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_hw_ta_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY   
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_na_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY 
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_na_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_ta_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_ta_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_na_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY 
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_na_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_ta_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_gld_sw_ta_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           0
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_na_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_na_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_ta_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_ta_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY   
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_na_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_na_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_ta_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_hw_ta_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY   
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_na_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY 
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_na_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_ta_nb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_ta_tb (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_na_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY 
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_na_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_ta_nb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}

__global__ void dgemm_main_tex_sw_ta_tb_fulltile (struct cublasDgemmParams parms)
{
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  FAST_IMUL
#undef  USE_TEX
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define FAST_IMUL         0
#define USE_TEX           1
#include "dgemm.h"
}
