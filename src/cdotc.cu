#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function cdotc */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float2> texX;
texture<float2> texY;

__global__ void cdotc_gld_main (struct cublasCdotcParams parms);
__global__ void cdotc_tex_main (struct cublasCdotcParams parms);

/*
 * hipComplex 
 * hipblasCdotc (int n, const hipComplex *x, int incx, const hipComplex *y, 
 *              int incy)
 *
 * computes the dot product of two single-complex vectors. It returns the 
 * dot product of the single-complex vectors x and y if successful, and complex
 * zero otherwise. It computes the sum for i = 0 to n - 1 of x[lx + i * incx] *
 * y[ly + i * incy], where lx = 1 if incx >= 0, else lx = 1 + (1 - n) * incx; 
 * ly is defined in a similar way using incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single-complex vector with n elements
 * incx   storage spacing between elements of x
 * y      single-complex vector with n elements
 * incy   storage spacing between elements of y
 *
 * Output
 * ------
 * returns single-complex dot product (zero if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/cdotc.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has nor been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to execute on GPU
 */
__host__ hipComplex CUBLASAPI hipblasCdotc (int n, const hipComplex *x, int incx, 
                                          const hipComplex *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCdotcParams params;
    hipError_t cudaStat;
    hipblasStatus_t status;
    hipComplex *devPtrT;
    int nbrCtas;
    int threadsPerCta;
    hipComplex *tx;
    hipComplex dot = make_hipComplex (0.0f, 0.0f);
    int i;
    int sizeX = n * (imax (1, abs(incx)));
    int sizeY = n * (imax (1, abs(incy)));
    size_t texXOfs = 0;
    size_t texYOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return dot;
    }
    
    if (n < CUBLAS_CDOTC_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_CDOTC_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_CDOTC_CTAS;
         threadsPerCta = CUBLAS_CDOTC_THREAD_COUNT;
    }

    /* early out if nothing to do */
    if (n <= 0) {
        return dot;
    }
    useTexture = ((sizeX < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeY < CUBLAS_MAX_1DBUF_SIZE));

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 50000) || /* experimental bound */
        ((sizeX == n) && (sizeY == n) && 
         (!(((uintptr_t) x) % CUBLAS_LONG_ALIGN)) && 
         (!(((uintptr_t) y) % CUBLAS_LONG_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        if ((cudaStat=hipBindTexture (&texYOfs,texY,y,sizeY*sizeof(y[0]))) !=
            hipSuccess) {
            hipUnbindTexture (texX);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return dot;
        }
        texXOfs /= sizeof(x[0]);
        texYOfs /= sizeof(y[0]);
    }

    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return dot;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (hipComplex *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return dot;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.cx = x;
    params.incx = incx;
    params.cy = y;
    params.incy = incy;
    params.result = devPtrT;
    params.texXOfs = (int)texXOfs;
    params.texYOfs = (int)texYOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        cdotc_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        cdotc_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }       
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return dot;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        return dot;
    }    

    for (i = 0; i < nbrCtas; i++) {
        dot = hipCaddf (dot, tx[i]);
    }

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texY)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }

    return dot;
}

__shared__ hipComplex partialSum[CUBLAS_CDOTU_THREAD_COUNT];        

__global__ void cdotc_gld_main (struct cublasCdotcParams parms) 
{
#undef  USE_TEX
#define USE_TEX 0
#include "cdotc.h"
}

__global__ void cdotc_tex_main (struct cublasCdotcParams parms) 
{
#undef  USE_TEX
#define USE_TEX 1
#include "cdotc.h"
}
