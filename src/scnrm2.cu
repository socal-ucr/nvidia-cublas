#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function scnrm2 */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void scnrm2_main (struct cublasScnrm2Params parms);

#define CUBLAS_SNRM2_STATE_ZERO    0
#define CUBLAS_SNRM2_STATE_TINY    1
#define CUBLAS_SNRM2_STATE_NORMAL  2
#define CUBLAS_SNRM2_STATE_HUGE    3
#define CUBLAS_SNRM2_STATE_DONE    4
__host__ static float local_snrm2 (int n, const float *sx, int incx);

__host__ static float local_snrm2 (int n, const float *sx, int incx)
{
    float cutLo = 4.441e-16f;
    float cutHi = 1.304e+19f;
    int i, ns, state;
    volatile float sum = 0.0f;
    volatile float hiTest;
    volatile float t = 0.0f;
    volatile float ta = 0.0f;
    volatile float xmax = 0.0f;
    volatile float xmaxRecip;

    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }
    ns = n * incx;
    hiTest = cutHi / (float)n;
    i = 0;
    state = CUBLAS_SNRM2_STATE_ZERO;
    while (state != CUBLAS_SNRM2_STATE_DONE) {
        switch (state) {
        case CUBLAS_SNRM2_STATE_ZERO:
            while ((i < ns) && ((t = sx[i]) == 0.0f)) {
                i += incx;
            }
            if (i >= ns) {
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {
                state = CUBLAS_SNRM2_STATE_TINY;
            }
            break;            
        case CUBLAS_SNRM2_STATE_TINY:
            xmax = (float)fabs(t);
            xmaxRecip = 1.0f / xmax;
            while ((i < ns) && ((ta = (float)fabs(t = sx[i])) < cutLo)) {
                if (ta > xmax) {
                    /* Adjust scale factor */
                    t = xmax / t;
                    sum = 1.0f + sum * t * t;
                    xmax = ta;
                    xmaxRecip = 1.0f / xmax;
                } else {
                    t = t * xmaxRecip;
                    sum += t * t;
                }
                i += incx; 
            }
            if (i >= ns) {
                sum = (float)sqrt(sum);
                sum = xmax * sum;
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {            
                state = CUBLAS_SNRM2_STATE_NORMAL;
            }
            break;
        case CUBLAS_SNRM2_STATE_NORMAL:
            sum = (sum * xmax) * xmax;
            while ((i < ns) && ((ta = (float)fabs(t = sx[i])) < hiTest)) {
                sum += t * t;
                i += incx;  
            }
            if (i >= ns) {
                sum = (float)sqrt(sum);
                state = CUBLAS_SNRM2_STATE_DONE;
            } else {
                state = CUBLAS_SNRM2_STATE_HUGE;
            }
            break;
        case CUBLAS_SNRM2_STATE_HUGE:
            xmax = ta;
            xmaxRecip = 1.0f / xmax;
            sum = (sum * xmaxRecip) * xmaxRecip;
            while (i < ns) {
                t = sx[i];
                ta = (float)fabs(t);
                if (ta > xmax) {
                    /* Adjust scale factor */
                    t = xmax / t;
                    sum = 1.0f + sum * t * t;
                    xmax = ta;
                    xmaxRecip = 1.0f / xmax;
                } else {
                    t = t * xmaxRecip;
                    sum += t * t;
                }
                i += incx; 
            }
            sum = (float)sqrt (sum);
            sum = xmax * sum;
            state = CUBLAS_SNRM2_STATE_DONE;
            break;
        }
    }
    return sum;
}

/*
 * float 
 * hipblasScnrm2 (int n, const hipComplex *x, int incx)
 *
 * computes the Euclidean norm of the single-complex n-vector x. This code 
 * uses simple scaling to avoid intermediate underflow and overflow.
 *
 * Input
 * -----
 * n      number of elements in input vector
 * x      single-complex vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * returns Euclidian norm (0 if n <= 0 or incx <= 0, or if an error occurs)
 *
 * Reference: http://www.netlib.org/blas/scnrm2.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 * 
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ float CUBLASAPI hipblasScnrm2 (int n, const hipComplex *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasScnrm2Params params;
    float *devPtrT;
    hipblasStatus_t status;
    hipError_t cudaStat;
    int nbrCtas;
    int threadsPerCta;
    float sum = 0.0f;
    float *tx;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return sum;
    }

    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return sum;
    }

    if (n < CUBLAS_SCNRM2_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_SCNRM2_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_SCNRM2_CTAS;
         threadsPerCta = CUBLAS_SCNRM2_THREAD_COUNT;
    }

    /* allocate memory to collect results, one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tx[0]), (void**)&devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return sum;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tx = (float *) calloc (nbrCtas, sizeof(tx[0]));
    if (!tx) {
        cublasFree (devPtrT);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return sum;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.incx = incx;
    params.result = devPtrT;

    cudaStat = hipGetLastError(); /* clear error status */
    scnrm2_main<<<nbrCtas,threadsPerCta>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        return sum;
    }

    /* Sum the results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tx[0]), devPtrT, 1, tx, 1);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrT);
        free (tx);
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
    }    
    
    sum = local_snrm2 (nbrCtas, tx, 1);

    status = cublasFree (devPtrT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tx);
    return sum;   
}

__global__ void scnrm2_main (struct cublasScnrm2Params parms) 
{
    float sum, scale, xmaxRecip;
    int i, state, ctaStart, totalThreads; 
    int n, tid;
    float t = 0.0f;
    float ta = 0.0f;
    float xmax = 0.0f;
    int ns;
    int totalIncx;
    const hipComplex *cx;
    __shared__ float partialSum[CUBLAS_SCNRM2_THREAD_COUNT];
   
    /* NOTE: wrapper must ensure that parms.n > 0 and parms.incx > 0 */
    tid = threadIdx.x;
    n = parms.n;
    cx = parms.cx;
    totalThreads = gridDim.x * CUBLAS_SCNRM2_THREAD_COUNT;
    ctaStart = CUBLAS_SCNRM2_THREAD_COUNT * blockIdx.x;
    ns = n * parms.incx;
    totalIncx = totalThreads * parms.incx;

    scale = 0.0f;
    sum = 1.0f;
    i = (ctaStart + tid) * parms.incx;
    while (i < ns) {
        if ((ta = fabsf(hipCrealf (cx[i]))) != 0.0f) {
            if (scale < ta) {
                t = scale / ta;
                sum = 1.0f + sum * t * t;
                scale = ta;
            } else {
                t = ta / scale;
                sum = sum + t * t;
            }
        }
        if ((ta = fabsf(hipCimagf (cx[i]))) != 0.0f) {
            if (scale < ta) {
                t = scale / ta;
                sum = 1.0f + sum * t * t;
                scale = ta;
            } else {
                t = ta / scale;
                sum = sum + t * t;
            }
        }            
        i += totalIncx;
    }
    partialSum[tid] = scale * sqrtf (sum);
    
    /*
     * FIXME: Because of the relatively complex state machine needed
     * to prevent overflow and underflow, right now we don't implement
     * a binary reduction tree but use a simple loop instead. Obviously
     * lower performance
     */

    __syncthreads();

    float cutLo = 4.441e-16f;
    float cutHi = 1.304e+19f;
    float hiTest;
    hiTest = cutHi / (float)n;
    
    /* let thread 0 sum the partial dot products for this CTA */
    if (tid == 0) {
        int nbrSums = CUBLAS_SNRM2_THREAD_COUNT;
        i = 0;
        state = CUBLAS_SNRM2_STATE_ZERO;
        while (state != CUBLAS_SNRM2_STATE_DONE) {
            /* we'd like a switch statement here */
            if (state == CUBLAS_SNRM2_STATE_ZERO) {
                sum = 0.0f;
                while (i < nbrSums) {
                    if (!((t = partialSum[i]) == 0.0f)) {
                        break;
                    }
                    i++;
                }
                state = (i >= nbrSums) ? CUBLAS_SNRM2_STATE_DONE : 
                                         CUBLAS_SNRM2_STATE_TINY;
                continue;
            }
            if (state == CUBLAS_SNRM2_STATE_TINY) {
                xmax = fabsf(t);
                xmaxRecip = 1.0f / xmax;
                while (i < nbrSums) {
                    if (!((ta = fabsf(t = partialSum[i])) < cutLo)) {
                        break;
                    }
                    if (ta > xmax) {
                        /* Adjust scale factor */
                        t = xmax / t;
                        sum = 1.0f + sum * t * t;
                        xmax = ta;
                        xmaxRecip = 1.0f / xmax;
                    } else {
                        t = t * xmaxRecip;
                        sum += t * t;
                    }
                    i++; 
                }
                if (i >= nbrSums) {
                    sum = xmax * sqrtf(sum);
                    state = CUBLAS_SNRM2_STATE_DONE;
                } else {            
                    state = CUBLAS_SNRM2_STATE_NORMAL;
                }
                continue;
            }
            if (state == CUBLAS_SNRM2_STATE_NORMAL) {
                sum = (sum * xmax) * xmax;
                while (i < nbrSums) {
                    if (!((ta = fabsf(t = partialSum[i])) < hiTest)) {
                        break;
                    }
                    sum += t * t;
                    i++;  
                }
                if (i >= nbrSums) {
                    sum = sqrtf(sum);
                    state = CUBLAS_SNRM2_STATE_DONE;
                } else {
                    state = CUBLAS_SNRM2_STATE_HUGE;
                }
                continue;
            }
            if (state == CUBLAS_SNRM2_STATE_HUGE) {
                xmax = ta;
                xmaxRecip = 1.0f / xmax;
                sum = (sum * xmaxRecip) * xmaxRecip;
                while (i < nbrSums) {
                    t = partialSum[i];
                    ta = fabsf(t);
                    if (ta > xmax) {
                        /* Adjust scale factor */
                        t = xmax / t;
                        sum = 1.0f + sum * t * t;
                        xmax = ta;
                        xmaxRecip = 1.0f / xmax;
                    } else {
                        t = t * xmaxRecip;
                        sum += t * t;
                    }
                    i++; 
                }
                sum = xmax * sqrtf (sum);
                state = CUBLAS_SNRM2_STATE_DONE;
                continue;
            }
        }
        parms.result[blockIdx.x] = sum;
    }
}

