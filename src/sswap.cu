#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function sswap */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;
texture<float> texY;

__global__ void sswap_gld_main (struct cublasSswapParams parms);
__global__ void sswap_tex_main (struct cublasSswapParams parms);

/*
 * void 
 * hipblasSswap (int n, float *x, int incx, float *y, int incy)
 *
 * interchanges single-precision vector x with single precision vector y..
 * For i = 0 to n - 1, it interchanges x[lx + i * incx] with y[ly + i *
 * incy], where lx = 1 if incx >= 0, else lx = 1 + (1 - n) * incx, and ly 
 * is defined in a similar way using incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * x      single-precision vector with n elements
 * incx   storage spacing between elements of x
 * y      single-precision vector with n elements
 * incy   storage spacing between elements of y
 *
 * Output
 * ------
 * x      input vector y (unchanged if n <= 0)
 * y      input vector x (unchanged if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/sswap.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status 
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSswap (int n, float *x, int incx, float *y,
                                     int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSswapParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;
    int sizeX = n * (imax (1, abs(incx)));
    int sizeY = n * (imax (1, abs(incy)));
    size_t texXOfs = 0;
    size_t texYOfs = 0;
    int useTexture;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }
    /* early out if nothing to do */
    if (n <= 0) {
        return;
    }

    useTexture = ((sizeX < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeY < CUBLAS_MAX_1DBUF_SIZE));

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 50000) || /* experimental bound */
        ((sizeX == n) && (sizeY == n) && 
         (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)) && 
         (!(((uintptr_t) y) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        if ((cudaStat=hipBindTexture (&texYOfs,texY,y,sizeY*sizeof(y[0]))) !=
            hipSuccess) {
            hipUnbindTexture (texX);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texXOfs /= sizeof(x[0]);
        texYOfs /= sizeof(y[0]);
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.incx = incx;
    params.sy = y;
    params.incy = incy;
    params.texXOfs = (int)texXOfs;
    params.texYOfs = (int)texYOfs;

    cublasVectorSplay (n, CUBLAS_SSWAP_THREAD_MIN, CUBLAS_SSWAP_THREAD_MAX,
                       CUBLAS_SSWAP_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        sswap_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        sswap_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }

    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texY)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}

__global__ void sswap_gld_main (struct cublasSswapParams parms)
{
#undef  USE_TEX
#define USE_TEX 0
#include "sswap.h"
}

__global__ void sswap_tex_main (struct cublasSswapParams parms)
{
#undef  USE_TEX
#define USE_TEX 1
#include "sswap.h"
}
