#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function stbsv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void stbsv_main_up_tr (struct cublasStbsvParams parms);
__global__ void stbsv_main_lo_tr (struct cublasStbsvParams parms);
__global__ void stbsv_main_up_nt (struct cublasStbsvParams parms);
__global__ void stbsv_main_lo_nt (struct cublasStbsvParams parms);

/*
 * void hipblasStbsv (char uplo, char trans, char diag, int n, int k,
 *                   const float *A, int lda, float *X, int incx)
 *
 * solves one of the systems of equations op(A)*x = b, where op(A) is either 
 * op(A) = A or op(A) = transpose(A). b and x are n element vectors, and A is
 * an n x n unit or non-unit, upper or lower triangular band matrix with k + 1
 * diagonals. No test for singularity or near-singularity is included in this
 * function. Such tests must be performed before calling this function.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix is an upper or lower triangular band 
 *        matrix as follows: If uplo == 'U' or 'u', A is an upper triangular
 *        band matrix. If uplo == 'L' or 'l', A is a lower triangular band
 *        matrix.
 * trans  specifies op(A). If trans == 'N' or 'n', op(A) = A. If trans == 'T',
 *        't', 'C', or 'c', op(A) = transpose(A).
 * diag   specifies whether A is unit triangular. If diag == 'U' or 'u', A is
 *        assumed to be unit triangular; thas is, diagonal elements are not
 *        read and are assumed to be unity. If diag == 'N' or 'n', A is not
 *        assumed to be unit triangular.
 * n      specifies the number of rows and columns of the matrix A. n must be
 *        at least zero.
 * k      specifies the number of super- or sub-diagonals. If uplo == 'U' or
 *        'u', k specifies the number of super-diagonals. If uplo == 'L' or
 *        'l', k specifies the number of sub-diagonals. k must at least be 
 *        zero.
 * A      single precision array of dimension (lda, n). If uplo == 'U' or 'u',
 *        the leading (k + 1) x n part of the array A must contain the upper
 *        triangular band matrix, supplied column by column, with the leading
 *        diagonal of the matrix in row (k + 1) of the array, the first super-
 *        diagonal starting at position 2 in row k, and so on. The top left 
 *        k x k triangle of the array A is not referenced. If uplo == 'L' or 
 *        'l', the leading (k + 1) x n part of the array A must constain the
 *        lower triangular band matrix, supplied column by column, with the
 *        leading diagonal of the matrix in row 1 of the array, the first
 *        sub-diagonal starting at position 1 in row 2, and so on. The bottom
 *        right k x k triangle of the array is not referenced.
 * x      single precision array of length at least (1+(n-1)*abs(incx)).
 * incx   storage spacing between elements of x. It must not be zero.
 *
 * Output
 * ------
 * x      updated to contain the solution vector x that solves op(A) * x = b.
 *
 * Reference: http://www.netlib.org/blas/stbsv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx == 0, n < 0, or n > 4070
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasStbsv (char uplo, char trans, char diag, int n,
                                     int k, const float *A, int lda, float *x,
                                     int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasStbsvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check other inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if ((toupper (trans) != 'N') && 
             (toupper (trans) != 'T') && 
             (toupper (trans) != 'C')) {
        info = 2;
    } 
    else if ((toupper (diag) != 'U') &&
             (toupper (diag) != 'N')) {
        info = 3;
    }
    /* FIXME: There should be no upper limit on n */
    else if ((n < 0) || (n > CUBLAS_STBSV_MAX_DIM)) {
        info = 4;
    }
    else if (k < 0) {
        info = 5;
    }
    else if (lda < (k + 1)) {
        info = 7;
    }
    else if (incx == 0) {
        info = 9;
    }
    if (info) {
        cublasXerbla ("STBSV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if (n == 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.k = k;
    params.n = n;
    params.A = A;
    params.x = x;
    params.lda = lda;
    params.incx = incx;
    params.trans = ((toupper(trans) == 'T') || (toupper(trans) == 'C'));
    params.unit = (toupper(diag) == 'U');
    params.up = (toupper(uplo) == 'U');

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.up) {
        if (params.trans) {
            stbsv_main_up_tr<<<CUBLAS_STBSV_CTAS,CUBLAS_STBSV_THREAD_COUNT>>>(params);
        } else {
            stbsv_main_up_nt<<<CUBLAS_STBSV_CTAS,CUBLAS_STBSV_THREAD_COUNT>>>(params);
        }
    } else {
         if (params.trans) {
            stbsv_main_lo_tr<<<CUBLAS_STBSV_CTAS,CUBLAS_STBSV_THREAD_COUNT>>>(params);
        } else {
            stbsv_main_lo_nt<<<CUBLAS_STBSV_CTAS,CUBLAS_STBSV_THREAD_COUNT>>>(params);
        }
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#define IDXX(i)             (startx + ((i) * parms.incx))
#define XINC                (CUBLAS_STBSV_THREAD_COUNT)

__shared__ float XX[CUBLAS_STBSV_MAX_DIM];  /* cached portion of vector x */
__shared__ float temp;                      /* current diagonal element */


__global__ void stbsv_main_up_tr (struct cublasStbsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    1
#include "stbsv.h"                
}

__global__ void stbsv_main_lo_tr (struct cublasStbsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    0
#include "stbsv.h"                
}

__global__ void stbsv_main_up_nt (struct cublasStbsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    1
#include "stbsv.h"                
}

__global__ void stbsv_main_lo_nt (struct cublasStbsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    0
#include "stbsv.h"                
}


