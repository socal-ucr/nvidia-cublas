#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function srotg */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

/*
 * void 
 * hipblasCrotg (hipComplex *ca, hipComplex *cb, float *sc, float *cs)
 *
 * constructs the complex Givens tranformation
 *
 *        ( sc  cs )
 *    G = (        ) ,  sc^2 + cabs(cs)^2 = 1,
 *        (-cs  sc )
 * 
 * which zeros the second entry of the complex 2-vector transpose(ca, cb).
 *
 * The quantity ca/cabs(ca)*norm(ca,cb) overwrites ca in storage. The 
 * function crot (n, x, incx, y, incy, sc, cs) is normally called next
 * to apply the transformation to a 2 x n matrix.
 *
 * Input
 * -----
 * ca     single-precision complex precision scalar
 * cb     single-precision complex scalar
 *
 * Output
 * ------
 * ca     single-precision complex ca/cabs(ca)*norm(ca,cb)
 * sc     single-precision cosine component of rotation matrix
 * cs     single-precision complex sine component of rotation matrix
 *
 * Reference: http://www.netlib.org/blas/crotg.f
 *
 * This function does not set any error status.
 */
__host__ void CUBLASAPI hipblasCrotg (hipComplex *pca, hipComplex cb, float *psc,
                                     hipComplex *pcs)
{
    hipComplex ca = *pca;
    float sc = *psc;
    hipComplex cs = *pcs;

    if (hipCabsf(ca) == 0.0f) {
        sc = 0.0f;
        cs = make_hipComplex (1.0f, 0.0f);
        ca = cb;
    } else {
        hipComplex alpha;
        float norm, scale;
        hipComplex tempA, tempB;
        /* Use scale factor to avoid intermediate underflow / overflow */
        scale = hipCabsf(ca) + hipCabsf(cb);
        tempA.x = ca.x / scale;
        tempA.y = ca.y / scale;
        tempB.x = cb.x / scale;
        tempB.y = cb.y / scale;
        norm = scale * sqrt (hipCabsf(tempA) * hipCabsf(tempA) + 
                             hipCabsf(tempB) * hipCabsf(tempB));
        alpha = ca;
        alpha.x = alpha.x / hipCabsf(ca);
        alpha.y = alpha.y / hipCabsf(ca);
        sc = hipCabsf(ca) / norm;
        cs = hipCmulf (alpha, hipConjf(cb));
        cs.x = cs.x / norm;
        cs.y = cs.y / norm;
        ca.x = alpha.x * norm;
        ca.y = alpha.y * norm;
    }
    *pca = ca;
    *psc = sc;
    *pcs = cs;
}
