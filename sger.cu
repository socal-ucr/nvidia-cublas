#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function sger */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

#define TILE_DIM_LOG        (5)
#define TILE_DIM            (1 << TILE_DIM_LOG)
#define TILE_SIZE           ((TILE_DIM) * (TILE_DIM))
#define ELEMS_PER_THREAD    ((TILE_SIZE)/(CUBLAS_SGER_THREAD_COUNT))
#define SUP_TILE_DIM        (TILE_DIM*CUBLAS_SGER_GRIDH)
#define A_NBR_COLS          (CUBLAS_SGER_THREAD_COUNT/TILE_DIM)

#if (TILE_DIM & (TILE_DIM - 1))
#error tile dimension must be a power of two
#endif

#if (CUBLAS_SGER_THREAD_COUNT < TILE_DIM)
#error thread count must be greater than or equal to tile dimension
#endif

#if ((TILE_SIZE)%CUBLAS_SGER_THREAD_COUNT)
#error number of tile elements must be integral multiple of thread count
#endif

#if (CUBLAS_SGER_THREAD_COUNT%TILE_DIM)
#error thread count must be integral multple of tile dimension
#endif

__global__ void sger_main_hw (struct cublasSgerParams parms);
__global__ void sger_main_fast_hw (struct cublasSgerParams parms);
__global__ void sger_main_sw (struct cublasSgerParams parms);

/*
 * hipblasSger (int m, int n, float alpha, const float *x, int incx, 
 *             const float *y, int incy, float *A, int lda)
 *
 * performs the symmetric rank 1 operation
 *
 *    A = alpha * x * transpose(y) + A,
 *
 * where alpha is a single precision scalar, x is an m element single 
 * precision vector, y is an n element single precision vector, and A 
 * is an m by n matrix consisting of single precision elements. Matrix A
 * is stored in column major format, and lda is the leading dimension of
 * the two-dimensional array used to store A.
 * 
 * Input
 * -----
 * m      specifies the number of rows of the matrix A. It must be at least 
 *        zero.
 * n      specifies the number of columns of the matrix A. It must be at 
 *        least zero.
 * alpha  single precision scalar multiplier applied to x * transpose(y)
 * x      single precision array of length at least (1 + (m - 1) * abs(incx))
 * incx   specifies the storage spacing between elements of x. incx must not
 *        be zero.
 * y      single precision array of length at least (1 + (n - 1) * abs(incy))
 * incy   specifies the storage spacing between elements of y. incy must not 
 *        be zero.
 * A      single precision array of dimensions (lda, n).
 * lda    leading dimension of two-dimensional array used to store matrix A
 *
 * Output
 * ------
 * A      updated according to A = alpha * x * transpose(y) + A
 *
 * Reference: http://www.netlib.org/blas/sger.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 * 
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if n < 0, incx == 0, incy == 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasSger (int m, int n, float alpha, const float *x,
                                    int incx, const float *y, int incy,
                                    float *A, int lda)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasSgerParams params;
    hipError_t cudaStat;
    int info = 0;
    int usePureHwStepper;
    int useFastImul;
    dim3 ctaDimsHw (((m+TILE_DIM-1)/TILE_DIM), ((n+TILE_DIM-1)/TILE_DIM));
    dim3 ctaDimsSw (CUBLAS_SGER_GRIDW, CUBLAS_SGER_GRIDH);

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    info = 0;
    if (m < 0) {
        info = 1;
    }
    else if (n < 0) {
        info = 2;
    }
    else if (incx == 0) {
        info = 5;
    }
     else if (incy == 0) {
        info = 7;
    }
    else if (lda < imax (1, m)) {
        info = 9;
    }
    if (info) {
        cublasXerbla ("SGER  ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((m == 0)|| (n == 0) || (alpha == 0.0f)) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.m = m;
    params.n = n;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    params.y = y;
    params.incy = incy;

    /* choose version using 24-bit multiplies if all dimensions are less than
     * 2001, so we can guarantee that no multiplication result exceeds (2000 *
     * 2000 * 4) < 2^24. Increments must be positive since we use unsigned
     * multiplication
     */
    useFastImul = ((params.lda  <= CUBLAS_FASTIMUL_F_MAX_DIM) && 
                   (params.m    <= CUBLAS_FASTIMUL_F_MAX_DIM) && 
                   (params.n    <= CUBLAS_FASTIMUL_F_MAX_DIM) &&
                   (params.incx <= CUBLAS_FASTIMUL_F_MAX_DIM) &&
                   (params.incy <= CUBLAS_FASTIMUL_F_MAX_DIM) &&
                   (params.incx > 0) && (params.incy > 0));

    /* choose HW-only stepping if dimensions of result matrix do not exceed the
     * maximum CTA grid dimensions.
     */
    usePureHwStepper = ((m < (CUBLAS_CTA_MAX_DIM * TILE_DIM)) &&
                        (n < (CUBLAS_CTA_MAX_DIM * TILE_DIM)));
    
    cudaStat = hipGetLastError(); /* clear error status */
    if (usePureHwStepper) {
        if (useFastImul) {
            sger_main_fast_hw<<<ctaDimsHw,CUBLAS_SGER_THREAD_COUNT>>>(params);
        } else {
            sger_main_hw<<<ctaDimsHw,CUBLAS_SGER_THREAD_COUNT>>>(params);
        }
    } else {
        sger_main_sw<<<ctaDimsSw,CUBLAS_SGER_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

__shared__ float xi[TILE_DIM];
__shared__ float yj[TILE_DIM];

__global__ void sger_main_hw (struct cublasSgerParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  FAST_IMUL
#define USE_MIXED_STEPPER 0
#define FAST_IMUL         0
#include "sger.h"
}

__global__ void sger_main_fast_hw (struct cublasSgerParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  FAST_IMUL
#define USE_MIXED_STEPPER 0
#define FAST_IMUL         1
#include "sger.h"
}

__global__ void sger_main_sw (struct cublasSgerParams parms) 
{
#undef  USE_MIXED_STEPPER
#undef  FAST_IMUL
#define USE_MIXED_STEPPER 1
#define FAST_IMUL         0
#include "sger.h"
}
