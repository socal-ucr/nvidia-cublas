#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function isamin */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

texture<float> texX;

__global__ void isamin_gld_main (struct cublasIsaminParams parms);
__global__ void isamin_tex_main (struct cublasIsaminParams parms);

/*
 * int 
 * isamin (int n, const float *x, int incx)
 *
 * finds the smallest index of the minimum magnitude element of single
 * precision vector x; that is, the result is the first i, i = 0 to n - 1, 
 * that minimizes abs(x[1 + i * incx])).
 * 
 * Input
 * -----
 * n      number of elements in input vector
 * x      single precision vector with n elements
 * incx   storage spacing between elements of x
 *
 * Output
 * ------
 * returns the smallest index (0 if n <= 0 or incx <= 0)
 *
 * Reference: http://www.netlib.org/scilib/blass.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ int CUBLASAPI hipblasIsamin (int n, const float *x, int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasIsaminParams params;
    float *devPtrTmx;
    int *devPtrTix;
    hipblasStatus_t status;
    hipError_t cudaStat;
    int nbrCtas;
    int threadsPerCta;
    int idx = 0;
    int *tix;
    float *tmx;
    int i, jmin;
    float smin, xabs;
    volatile union {
        float f;
        int i;
    } cvt;
    int sizeX = n * (imax (1, abs(incx)));
    size_t texXOfs = 0;
    int useTexture;    

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return idx;
    }

    /* early out if nothing to do */
    if ((n <= 0) || (incx <= 0)) {
        return idx;
    }

    if (n < CUBLAS_ISAMIN_CTAS) {
         nbrCtas = n;
         threadsPerCta = CUBLAS_ISAMIN_THREAD_COUNT;
    } else {
         nbrCtas = CUBLAS_ISAMIN_CTAS;
         threadsPerCta = CUBLAS_ISAMIN_THREAD_COUNT;
    }

    useTexture = sizeX < CUBLAS_MAX_1DBUF_SIZE;

    /* Currently, the overhead for using textures is high. Do not use texture
     * for vectors that are short, or those that are aligned and have unit
     * stride and thus have nicely coalescing GLDs.
     */
    if ((n < 100000) || /* experimental bound */
        ((sizeX == n) && (!(((uintptr_t) x) % CUBLAS_WORD_ALIGN)))) {
        useTexture = 0;
    }

    if (useTexture) {
        if ((cudaStat=hipBindTexture (&texXOfs,texX,x,sizeX*sizeof(x[0]))) !=
            hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return idx;
        }
        texXOfs /= sizeof(x[0]);
    }

    /* allocate memory to collect results (index, minimum), one per CTA */
    status = cublasAlloc (nbrCtas, sizeof(tmx[0]), (void**)&devPtrTmx);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, status);
        return idx;
    }
    status = cublasAlloc (nbrCtas, sizeof(tix[0]), (void**)&devPtrTix);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasFree (devPtrTmx);
        cublasSetError (ctx, status);
        return idx;
    }

    /* allocate small buffer to retrieve the per-CTA results */
    tmx = (float *) calloc (nbrCtas, sizeof(tmx[0]));
    if (!tmx) {
        cublasFree (devPtrTmx);
        cublasFree (devPtrTix);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return idx;
    }
    tix = (int *) calloc (nbrCtas, sizeof(tix[0]));
    if (!tix) {
        cublasFree (devPtrTmx);
        cublasFree (devPtrTix);
        free (tmx);
        cublasSetError (ctx, HIPBLAS_STATUS_ALLOC_FAILED);
        return idx;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.sx = x;
    params.incx = incx;
    params.resMin = devPtrTmx;
    params.resPos = devPtrTix;
    params.texXOfs = (int)texXOfs;

    cudaStat = hipGetLastError(); /* clear error status */
    if (useTexture) {
        isamin_tex_main<<<nbrCtas,threadsPerCta>>>(params);
    } else {
        isamin_gld_main<<<nbrCtas,threadsPerCta>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
        cublasFree (devPtrTmx);
        cublasFree (devPtrTix);
        free (tmx);
        free (tix);
        return idx;
    }

    /* Get idx/min results from each CTA */
    status = hipblasGetVector (nbrCtas, sizeof(tmx[0]), devPtrTmx, 1, tmx, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        cublasFree (devPtrTmx);
        cublasFree (devPtrTix);
        free (tmx);
        free (tix);
        return idx;
    }
    status = hipblasGetVector (nbrCtas, sizeof(tix[0]), devPtrTix, 1, tix, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        cublasFree (devPtrTmx);
        cublasFree (devPtrTix);
        free (tmx);
        free (tix);
        return idx;
    }

    /* find smallest index of minimum value among CTA results */
    cvt.i = 0x7f800000;  /* IEEE-754 single infinity */
    smin = cvt.f;
    for (i = 0; i < nbrCtas; i++) {
        xabs = tmx[i];
        jmin = tix[i];
        if ((xabs < smin) || ((xabs == smin) && (jmin < idx))) {
            idx = jmin;
            smin = xabs;
        }
    }
    
    /* translate result from 0-indexed to 1-indexed */
    idx++;

    status = cublasFree (devPtrTmx);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    status = cublasFree (devPtrTix);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR); /* corruption ? */
    }
    free (tmx);
    free (tix);
 
    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texX)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
   return idx;   
}

__shared__ int   partialIMin[CUBLAS_ISAMIN_THREAD_COUNT];
__shared__ float partialSMin[CUBLAS_ISAMIN_THREAD_COUNT];

__global__ void isamin_gld_main (struct cublasIsaminParams parms)
{
#undef  USE_TEX
#define USE_TEX 0
#include "isamin.h"
}

__global__ void isamin_tex_main (struct cublasIsaminParams parms)
{
#undef  USE_TEX
#define USE_TEX 1
#include "isamin.h"
}
