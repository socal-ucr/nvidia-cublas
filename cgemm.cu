#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-3 function cgemm */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

// dimension m, counter i
// dimension n, counter j
// dimension k, counter l

#if (CUBLAS_CGEMM_GRIDW!=CUBLAS_CGEMM_GRIDH)
#error super tile is not square!
#endif

/* Use square 16x16 tiles to access and cache portions of source matrices A,B 
 * and result matrix C
 */
#define TILE_DIM_LOG    (4)
#define TILE_DIM        (1 << TILE_DIM_LOG)
#define TILE_SIZE       (TILE_DIM*TILE_DIM)
#define SUP_TILE_DIM    (TILE_DIM*CUBLAS_CGEMM_GRIDW)

/* In cases where there are more tile elements than threads in a CTA, each
 * thread needs to walk through the tile. To keep the walking pattern simple,
 * we make sure that the number of threads is an integral multiple of the
 * number of elements (i.e. each thread deals with exactly the same number
 * of elements), and that tile dimension (number of rows / number of columns)
 * divides the thread count without remainder. After assigning an initial
 * element to each thread, the thread can then access further elements by 
 * remaining in the same tile row and merely stepping through columns that
 * are COL_INCR apart.
 */
#if ((TILE_SIZE%CUBLAS_CGEMM_THREAD_COUNT)!=0)
#error TILE_SIZE and THREAD_COUNT do not divide evenly!
#endif
#if ((CUBLAS_CGEMM_THREAD_COUNT%TILE_DIM)!=0)
#error THREAD_COUNT and TILE_DIM do not divide evenly!
#endif

#define COL_INCR               (CUBLAS_CGEMM_THREAD_COUNT/TILE_DIM)
#define C_ELEMS_PER_THREAD     (TILE_SIZE/CUBLAS_CGEMM_THREAD_COUNT)
#define A_ELEMS_PER_THREAD     (TILE_SIZE/CUBLAS_CGEMM_THREAD_COUNT)
#define B_ELEMS_PER_THREAD     (TILE_SIZE/CUBLAS_CGEMM_THREAD_COUNT)

__global__ void cgemm_1_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_sw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_1_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_sw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_sw_gld_fulltile (struct cublasCgemmParams parms);

__global__ void cgemm_1_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_hw_gld (struct cublasCgemmParams parms);
__global__ void cgemm_1_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_hw_gld_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_hw_gld_fulltile (struct cublasCgemmParams parms);

__global__ void cgemm_1_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_sw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_1_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_sw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_sw_tex_fulltile (struct cublasCgemmParams parms);

__global__ void cgemm_1_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_hw_tex (struct cublasCgemmParams parms);
__global__ void cgemm_1_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_2_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_3_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_4_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_5_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_6_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_7_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_8_main_hw_tex_fulltile (struct cublasCgemmParams parms);
__global__ void cgemm_9_main_hw_tex_fulltile (struct cublasCgemmParams parms);

texture<float2> texA;
texture<float2> texB;

__shared__ float AA_r[(TILE_DIM+1)*TILE_DIM]; 
__shared__ float BB_r[(TILE_DIM+1)*TILE_DIM]; 
__shared__ float AA_i[(TILE_DIM+1)*TILE_DIM]; 
__shared__ float BB_i[(TILE_DIM+1)*TILE_DIM];

typedef void (*pf) (struct cublasCgemmParams parms);

static pf cgemm_hw[64] = {
    cgemm_9_main_hw_gld, /* C = alpha*transpose(A)*transpose(B) + beta*C */
    cgemm_8_main_hw_gld, /* C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
    cgemm_7_main_hw_gld, /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
    cgemm_6_main_hw_gld, /* C = alpha*conjg(transpose(A))*conjg(transpose(B))+beta*C */
    cgemm_3_main_hw_gld, /* C = alpha*transpose(A)*B + beta*C */
    cgemm_3_main_hw_gld, /* C = alpha*transpose(A)*B + beta*C */
    cgemm_2_main_hw_gld, /* C = alpha*conj(transpose(A))*B + beta*C */
    cgemm_2_main_hw_gld, /* C = alpha*conj(transpose(A))*B + beta*C */
    cgemm_5_main_hw_gld, /* C = alpha*A*transpose(B) + beta*C */
    cgemm_4_main_hw_gld, /* C = alpha*A*conj(transpose(B)) + beta*C */
    cgemm_5_main_hw_gld, /* C = alpha*A*transpose(B) + beta*C */
    cgemm_4_main_hw_gld, /* C = alpha*A*conj(transpose(B)) + beta*C */
    cgemm_1_main_hw_gld, /* C = alpha*A*B + beta*C */
    cgemm_1_main_hw_gld, /* C = alpha*A*B + beta*C */
    cgemm_1_main_hw_gld, /* C = alpha*A*B + beta*C */
    cgemm_1_main_hw_gld, /* C = alpha*A*B + beta*C */
    cgemm_9_main_hw_gld_fulltile,
    cgemm_8_main_hw_gld_fulltile,
    cgemm_7_main_hw_gld_fulltile,
    cgemm_6_main_hw_gld_fulltile,
    cgemm_3_main_hw_gld_fulltile,
    cgemm_3_main_hw_gld_fulltile,
    cgemm_2_main_hw_gld_fulltile,
    cgemm_2_main_hw_gld_fulltile,
    cgemm_5_main_hw_gld_fulltile,
    cgemm_4_main_hw_gld_fulltile,
    cgemm_5_main_hw_gld_fulltile,
    cgemm_4_main_hw_gld_fulltile,
    cgemm_1_main_hw_gld_fulltile,
    cgemm_1_main_hw_gld_fulltile,
    cgemm_1_main_hw_gld_fulltile,
    cgemm_1_main_hw_gld_fulltile,
    cgemm_9_main_hw_tex,
    cgemm_8_main_hw_tex,
    cgemm_7_main_hw_tex,
    cgemm_6_main_hw_tex,
    cgemm_3_main_hw_tex,
    cgemm_3_main_hw_tex,
    cgemm_2_main_hw_tex,
    cgemm_2_main_hw_tex,
    cgemm_5_main_hw_tex,
    cgemm_4_main_hw_tex,
    cgemm_5_main_hw_tex,
    cgemm_4_main_hw_tex,
    cgemm_1_main_hw_tex,
    cgemm_1_main_hw_tex,
    cgemm_1_main_hw_tex,
    cgemm_1_main_hw_tex,
    cgemm_9_main_hw_tex_fulltile,
    cgemm_8_main_hw_tex_fulltile,
    cgemm_7_main_hw_tex_fulltile,
    cgemm_6_main_hw_tex_fulltile,
    cgemm_3_main_hw_tex_fulltile,
    cgemm_3_main_hw_tex_fulltile,
    cgemm_2_main_hw_tex_fulltile,
    cgemm_2_main_hw_tex_fulltile,
    cgemm_5_main_hw_tex_fulltile,
    cgemm_4_main_hw_tex_fulltile,
    cgemm_5_main_hw_tex_fulltile,
    cgemm_4_main_hw_tex_fulltile,
    cgemm_1_main_hw_tex_fulltile,
    cgemm_1_main_hw_tex_fulltile,
    cgemm_1_main_hw_tex_fulltile,
    cgemm_1_main_hw_tex_fulltile
};

static pf cgemm_sw[64] = {
    cgemm_9_main_sw_gld, /* C=alpha*transpose(A)*transpose(B) + beta*C */
    cgemm_8_main_hw_gld, /* C=alpha*transpose(A)*conjg(transpose(B)) + beta*C*/
    cgemm_7_main_sw_gld, /* C=alpha*conjg(transpose(A))*transpose(B) + beta*C*/
    cgemm_6_main_sw_gld, /* C=alpha*conjg(transpose(A))*conjg(transpose(B))+beta*C */
    cgemm_3_main_sw_gld, /* C=alpha*transpose(A)*B + beta*C */
    cgemm_3_main_sw_gld, /* C=alpha*transpose(A)*B + beta*C */
    cgemm_2_main_sw_gld, /* C=alpha*conj(transpose(A))*B + beta*C */
    cgemm_2_main_sw_gld, /* C=alpha*conj(transpose(A))*B + beta*C */
    cgemm_5_main_sw_gld, /* C=alpha*A*transpose(B) + beta*C */
    cgemm_4_main_sw_gld, /* C=alpha*A*conj(transpose(B)) + beta*C */
    cgemm_5_main_sw_gld, /* C=alpha*A*transpose(B) + beta*C */
    cgemm_4_main_sw_gld, /* C=alpha*A*conj(transpose(B)) + beta*C */
    cgemm_1_main_sw_gld, /* C=alpha*A*B + beta*C */
    cgemm_1_main_sw_gld, /* C=alpha*A*B + beta*C */
    cgemm_1_main_sw_gld, /* C=alpha*A*B + beta*C */
    cgemm_1_main_sw_gld, /* C=alpha*A*B + beta*C */
    cgemm_9_main_sw_gld_fulltile,
    cgemm_8_main_sw_gld_fulltile,
    cgemm_7_main_sw_gld_fulltile,
    cgemm_6_main_sw_gld_fulltile,
    cgemm_3_main_sw_gld_fulltile,
    cgemm_3_main_sw_gld_fulltile,
    cgemm_2_main_sw_gld_fulltile,
    cgemm_2_main_sw_gld_fulltile,
    cgemm_5_main_sw_gld_fulltile,
    cgemm_4_main_sw_gld_fulltile,
    cgemm_5_main_sw_gld_fulltile,
    cgemm_4_main_sw_gld_fulltile,
    cgemm_1_main_sw_gld_fulltile,
    cgemm_1_main_sw_gld_fulltile,
    cgemm_1_main_sw_gld_fulltile,
    cgemm_1_main_sw_gld_fulltile,
    cgemm_9_main_sw_tex,
    cgemm_8_main_sw_tex,
    cgemm_7_main_sw_tex,
    cgemm_6_main_sw_tex,
    cgemm_3_main_sw_tex,
    cgemm_3_main_sw_tex,
    cgemm_2_main_sw_tex,
    cgemm_2_main_sw_tex,
    cgemm_5_main_sw_tex,
    cgemm_4_main_sw_tex,
    cgemm_5_main_sw_tex,
    cgemm_4_main_sw_tex,
    cgemm_1_main_sw_tex,
    cgemm_1_main_sw_tex,
    cgemm_1_main_sw_tex,
    cgemm_1_main_sw_tex,
    cgemm_9_main_sw_tex_fulltile,
    cgemm_8_main_sw_tex_fulltile,
    cgemm_7_main_sw_tex_fulltile,
    cgemm_6_main_sw_tex_fulltile,
    cgemm_3_main_sw_tex_fulltile,
    cgemm_3_main_sw_tex_fulltile,
    cgemm_2_main_sw_tex_fulltile,
    cgemm_2_main_sw_tex_fulltile,
    cgemm_5_main_sw_tex_fulltile,
    cgemm_4_main_sw_tex_fulltile,
    cgemm_5_main_sw_tex_fulltile,
    cgemm_4_main_sw_tex_fulltile,
    cgemm_1_main_sw_tex_fulltile,
    cgemm_1_main_sw_tex_fulltile,
    cgemm_1_main_sw_tex_fulltile,
    cgemm_1_main_sw_tex_fulltile
};

/*
 * void hipblasCgemm (char transa, char transb, int m, int n, int k, 
 *                   hipComplex alpha, const hipComplex *A, int lda, 
 *                   const hipComplex *B, int ldb, hipComplex beta, 
 *                   hipComplex *C, int ldc)
 *
 * cgemm performs one of the matrix-matrix operations
 *
 *    C = alpha * op(A) * op(B) + beta*C,
 *
 * where op(X) is one of
 *
 *    op(X) = X   or   op(X) = transpose  or  op(X) = conjg(transpose(X))
 *
 * alpha and beta are single-complex scalars, and A, B and C are matrices
 * consisting of single-complex elements, with op(A) an m x k matrix, op(B)
 * a k x n matrix and C an m x n matrix.
 *
 * Input
 * -----
 * transa specifies op(A). If transa == 'N' or 'n', op(A) = A. If transa == 
 *        'T' or 't', op(A) = transpose(A). If transa == 'C' or 'c', op(A) = 
 *        conjg(transpose(A)).
 * transb specifies op(B). If transa == 'N' or 'n', op(B) = B. If transb == 
 *        'T' or 't', op(B) = transpose(B). If transb == 'C' or 'c', op(B) = 
 *        conjg(transpose(B)).
 * m      number of rows of matrix op(A) and rows of matrix C. It must be at
 *        least zero.
 * n      number of columns of matrix op(B) and number of columns of C. It 
 *        must be at least zero.
 * k      number of columns of matrix op(A) and number of rows of op(B). It 
 *        must be at least zero.
 * alpha  single-complex scalar multiplier applied to op(A)op(B)
 * A      single-complex array of dimensions (lda, k) if transa ==  'N' or 
 *        'n'), and of dimensions (lda, m) otherwise.
 * lda    leading dimension of A. When transa == 'N' or 'n', it must be at 
 *        least max(1, m) and at least max(1, k) otherwise.
 * B      single-complex array of dimensions (ldb, n) if transb == 'N' or 'n', 
 *        and of dimensions (ldb, k) otherwise
 * ldb    leading dimension of B. When transb == 'N' or 'n', it must be at 
 *        least max(1, k) and at least max(1, n) otherwise.
 * beta   single-complex scalar multiplier applied to C. If beta is zero, C 
 *        does not have to be a valid input.
 * C      single precision array of dimensions (ldc, n)
 * ldc    leading dimension of C. Must be at least max(1, m).
 *
 * Output
 * ------
 * C      updated according to C = alpha*op(A)*op(B) + beta*C
 *
 * Reference: http://www.netlib.org/blas/cgemm.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if any of m, n, or k are < 0
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasCgemm (char transa, char transb, int m, int n,
                                     int k, hipComplex alpha, 
                                     const hipComplex *A, int lda, 
                                     const hipComplex *B, int ldb,
                                     hipComplex beta, hipComplex *C, int ldc)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCgemmParams params;
    hipError_t cudaStat;
    size_t texAOfs = 0;
    size_t texBOfs = 0;
    int nrowa, nrowb;
    int notransa, notransb;
    int conja, conjb;
    int info = 0;
    int usePureHwStepper;
    int fullTilesOnly;
    int useFastImul;
    int useTexture;
    int funcIdx;
    int sizeA, sizeB;

    dim3 ctaDimsHw (((n+TILE_DIM-1)/TILE_DIM), ((m+TILE_DIM-1)/TILE_DIM));
    dim3 ctaDimsSw (CUBLAS_CGEMM_GRIDW, CUBLAS_CGEMM_GRIDH);

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    nrowa = (toupper(transa) == 'N') ? m : k;
    nrowb = (toupper(transb) == 'N') ? k : n;

    if ((toupper(transa) != 'N') && 
        (toupper(transa) != 'C') && 
        (toupper(transa) != 'T')) {
        info = 1;
    } 
    else if ((toupper(transb) != 'N') && 
             (toupper(transb) != 'C') && 
             (toupper(transb) != 'T')) {
        info = 2;
    }
    else if (m < 0) {
        info = 3;
    }
    else if (n < 0) {
        info = 4;
    }
    else if (k < 0) {
        info = 5;
    }
    else if (lda < imax(1, nrowa)) {
        info = 8;
    }
    else if (ldb < imax(1, nrowb)) {
        info = 10;
    }
    else if (ldc < imax(1, m)) {
        info = 13;
    }
    if (info) {
        cublasXerbla ("CGEMM ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if ((m == 0) || (n == 0) || 
        ((((hipCrealf(alpha) == 0.0f) && (hipCimagf(alpha) == 0.0f)) || (k == 0)) 
         && ((hipCrealf(beta) == 1.0f) && (hipCimagf(beta) == 0.0f)))) {
        return;
    }

    /* choose version using 24-bit multiplies if all dimensions are less than
     * 1410, so we can guarantee that no intra-matrix address exceeds (1410 *
     * 1410 * 8) < 2^24.
     */
    useFastImul =((lda < CUBLAS_FASTIMUL_D_MAX_DIM) && 
                  (ldb < CUBLAS_FASTIMUL_D_MAX_DIM) && 
                  (ldc < CUBLAS_FASTIMUL_D_MAX_DIM) &&
                  (m   < CUBLAS_FASTIMUL_D_MAX_DIM) && 
                  (n   < CUBLAS_FASTIMUL_D_MAX_DIM) && 
                  (k   < CUBLAS_FASTIMUL_D_MAX_DIM));

    if (useFastImul) {
        cublasFastCgemm (ctx, transa, transb, m, n, k, alpha, A, lda, B, ldb, 
                         beta, C, ldc);
        return;
    }        
   
    sizeA = lda * ((toupper(transa) == 'N') ? k : m);
    sizeB = ldb * ((toupper(transb) == 'N') ? n : k);   

    conja  = toupper(transa) == 'C';
    conjb  = toupper(transb) == 'C';
    notransa = toupper(transa) == 'N';
    notransb = toupper(transb) == 'N';

    /* We can only use texture if the matrices fit into the largest matrix 
     * size supported.
     */
    useTexture = ((sizeA < CUBLAS_MAX_1DBUF_SIZE) &&
                  (sizeB < CUBLAS_MAX_1DBUF_SIZE));

    /* choose HW-only stepping if dimensions of result matrix do not exceed the
     * maximum CTA grid dimensions.
     */
    usePureHwStepper = ((m < (CUBLAS_CTA_MAX_DIM * TILE_DIM)) &&
                        (n < (CUBLAS_CTA_MAX_DIM * TILE_DIM)));

    /* we can eliminate checking for endcases if we know all tiles are fully
     * populated. Important benchmark case!
     */
    fullTilesOnly = (((m % TILE_DIM) == 0) &&
                     ((n % TILE_DIM) == 0) &&
                     ((k % TILE_DIM) == 0));

    /* currently, texture binding is expensive, so using texture fetches
     * is a net negative for small cases. For matrices where each row is
     * aligned, GLD coalesces nicely and is faster, so don't use texture.
     */
    if ((!(((ptrdiff_t) A) % CUBLAS_WORD_ALIGN) && 
         !(((ptrdiff_t) B) % CUBLAS_WORD_ALIGN) &&
         !(lda % (CUBLAS_WORD_ALIGN / sizeof(A[0]))) &&
         !(ldb % (CUBLAS_WORD_ALIGN / sizeof(B[0]))))) {
        useTexture = 0;
    }
    
    if (useTexture){
        if ((cudaStat=hipBindTexture (&texAOfs,texA,A,sizeA*sizeof(A[0]))) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        if ((cudaStat=hipBindTexture (&texBOfs,texB,B,sizeB*sizeof(B[0]))) != hipSuccess) {
            hipUnbindTexture (texA);
            cublasSetError (ctx, HIPBLAS_STATUS_MAPPING_ERROR);
            return;
        }
        texAOfs /= sizeof(A[0]);
        texBOfs /= sizeof(B[0]);
    }

    memset (&params, 0, sizeof(params));
    params.m = m;
    params.n = n;
    params.k = k;
    params.alpha = alpha;
    params.A = A;
    params.lda = lda;
    params.B = B;
    params.ldb = ldb;
    params.beta = beta;
    params.C = C;
    params.ldc =ldc;
    params.texAOfs = (int)texAOfs;
    params.texBOfs = (int)texBOfs;
    
    funcIdx = ((useTexture << 5) | (fullTilesOnly << 4) | (notransa << 3) | 
               (notransb << 2) | (conja << 1) | (conjb << 0));

    cudaStat = hipGetLastError(); /* clear error status */
    if (usePureHwStepper) {
        cgemm_hw[funcIdx]<<<ctaDimsHw,CUBLAS_CGEMM_THREAD_COUNT>>>(params);
    } else {
        cgemm_sw[funcIdx]<<<ctaDimsSw,CUBLAS_CGEMM_THREAD_COUNT>>>(params);
    }
    cudaStat = hipGetLastError(); /* check for launch error */
    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
    if (useTexture) {
        if ((cudaStat = hipUnbindTexture (texA)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
        if ((cudaStat = hipUnbindTexture (texB)) != hipSuccess) {
            cublasSetError (ctx, HIPBLAS_STATUS_INTERNAL_ERROR);
        }
    }
}

__global__ void cgemm_1_main_sw_gld (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_sw_gld (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_sw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_hw_gld (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_hw_gld (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_hw_gld (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_sw_gld_fulltile (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_sw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_hw_gld_fulltile (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_hw_gld_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           0
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

/*************************************************************************************/

__global__ void cgemm_1_main_sw_tex (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_sw_tex (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_sw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_hw_tex (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_hw_tex (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_hw_tex (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   0
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_sw_tex_fulltile (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_sw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 1
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_1_main_hw_tex_fulltile (struct cublasCgemmParams parms) 
{
    /* C = alpha*A*B + beta*C. */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_2_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conj(transpose(A))*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_3_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*B + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            0
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_4_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*conj(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_5_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*A*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            0
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_6_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_7_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*conjg(transpose(A))*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            1
#define CONJGB            0
#include "cgemm.h"
}

__global__ void cgemm_8_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /*  C = alpha*transpose(A)*conjg(transpose(B)) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            1
#include "cgemm.h"
}

__global__ void cgemm_9_main_hw_tex_fulltile (struct cublasCgemmParams parms)
{
    /* C = alpha*transpose(A)*transpose(B) + beta*C */
#undef  USE_TEX
#undef  FAST_IMUL
#undef  FULL_TILES_ONLY
#undef  USE_MIXED_STEPPER
#undef  TRANSA
#undef  TRANSB
#undef  CONJGA
#undef  CONJGB
#define USE_TEX           1
#define FAST_IMUL         0
#define FULL_TILES_ONLY   1
#define USE_MIXED_STEPPER 0
#define TRANSA            1
#define TRANSB            1
#define CONJGA            0
#define CONJGB            0
#include "cgemm.h"
}
