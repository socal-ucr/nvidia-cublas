#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-1 function caxpy */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void caxpy_main (struct cublasCaxpyParams parms);

/*
 * void
 * hipblasCaxpy (int n, hipComplex alpha, const hipComplex *x, int incx, 
 *              hipComplex *y, int incy)
 *
 * multiplies complex vector x by complex scalar alpha and adds the 
 * result to complex vector y; that is, it overwrites complex y with 
 * complex alpha * x + y. For i = 0 to n - 1, it replaces y[ly + i * incy] 
 * with alpha * x[lx + i * incx] + y[ly + i * incy], where lx = 0 if incx 
 * >= 0, else lx = 1 + (1 - n) * incx, and ly is defined in a similar way 
 * using incy.
 *
 * Input
 * -----
 * n      number of elements in input vectors
 * alpha  complex scalar multiplier
 * x      complex vector with n elements
 * incx   storage spacing between elements of x
 * y      complex vector with n elements
 * incy   storage spacing between elements of y
 *
 * Output
 * ------
 * y      complex result (unchanged if n <= 0)
 *
 * Reference: http://www.netlib.org/blas/caxpy.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasCaxpy (int n, hipComplex alpha, 
                                     const hipComplex *x, int incx,
                                     hipComplex *y, int incy)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasCaxpyParams params;
    hipError_t cudaStat;
    int nbrCtas;
    int elemsPerCta;
    int threadsPerCta;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* early out if nothing to do */
    if ((n <= 0) || ((fabsf (hipCrealf(alpha)) + fabsf (hipCimagf(alpha))) == 0.0f)){
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n  = n;
    params.cx = x;
    params.ca = alpha;
    params.incx = incx;
    params.cy = y;
    params.incy = incy;

    cublasVectorSplay (n, CUBLAS_CAXPY_THREAD_MIN, CUBLAS_CAXPY_THREAD_MAX,
                       CUBLAS_CAXPY_CTAS_MAX, &nbrCtas, &elemsPerCta, 
                       &threadsPerCta);

    cudaStat = hipGetLastError(); /* clear error status */
    caxpy_main<<<nbrCtas,threadsPerCta>>>(params);
    cudaStat = hipGetLastError(); /* check for launch error */

    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

__global__ void caxpy_main (struct cublasCaxpyParams parms) 
{
    int i, n, tid, totalThreads, ctaStart;
    const hipComplex *cx;
    hipComplex *cy;

    /* NOTE: host wrapper must ensure that parms.n > 0  */

    tid = threadIdx.x;
    n = parms.n;
    cx = parms.cx;
    cy = parms.cy;
    totalThreads = gridDim.x * blockDim.x;
    ctaStart = blockDim.x * blockIdx.x;
    
    if (parms.incy == 0) {
        if ((blockIdx.x == 0) && (tid == 0)) {
            /* FIXME: This code is functionally correct, but inefficient */
            int ix = (parms.incx < 0) ? ((1 - parms.n) * parms.incx) : 0;
            hipComplex sum;
            sum.x = 0.0f;
            sum.y = 0.0f;
            for (i = 0; i < parms.n; i++) {
                sum = hipCaddf (sum, hipCmulf (parms.ca, cx[ix]));
                ix += parms.incx;
            }
            parms.cy[0] = hipCaddf (parms.cy[0], sum);
        }
    } else if ((parms.incx == parms.incy) && (parms.incx > 0)) {
        /* equal, positive, increments */
        if (parms.incx == 1) {
            /* both increments equal to 1 */
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                cy[i] = hipCaddf (cy[i], hipCmulf (parms.ca, cx[i]));
            }
        } else {
            /* equal, positive, non-unit increments. */
            for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
                cy[i*parms.incx] = hipCaddf (cy[i*parms.incx], 
                                           hipCmulf(parms.ca,cx[i*parms.incx]));
            }
        }
    } else {
        /* unequal or nonpositive increments */
        int ix = ((parms.incx < 0) ? ((1 - n) * parms.incx) : 0);
        int iy = ((parms.incy < 0) ? ((1 - n) * parms.incy) : 0);
        for (i = ctaStart + tid; i < parms.n; i += totalThreads) {
            cy[iy+i*parms.incy] = hipCaddf(cy[iy+i*parms.incy], 
                                         hipCmulf(parms.ca,cx[ix+i*parms.incx]));
        }
    }
}
