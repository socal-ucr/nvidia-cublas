#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  
 *
 * This software and the information contained herein is being provided 
 * under the terms and conditions of a Source Code License Agreement.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This file contains the implementation of the BLAS-2 function strsv */

#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <limits.h>
#include <ctype.h>
#include <math.h>
#include "hipblas.h"   /* CUBLAS public header file  */
#include "cublasP.h"  /* CUBLAS private header file */

__global__ void strsv_main_tr_up (struct cublasStrsvParams parms);
__global__ void strsv_main_tr_lo (struct cublasStrsvParams parms);
__global__ void strsv_main_nt_up (struct cublasStrsvParams parms);
__global__ void strsv_main_nt_lo (struct cublasStrsvParams parms);

/*
 * void 
 * hipblasStrsv (char uplo, char trans, char diag, int n, const float *A, 
 *              int lda, float *x, int incx)
 *
 * solves a system of equations op(A) * x = b, where op(A) is either A or 
 * transpose(A). b and x are single precision vectors consisting of n
 * elements, and A is an n x n matrix composed of a unit or non-unit, upper
 * or lower triangular matrix. Matrix A is stored in column major format,
 * and lda is the leading dimension of the two-diemnsional array containing
 * A.
 *
 * No test for singularity or near-singularity is included in this function. 
 * Such tests must be performed before calling this function.
 *
 * Input
 * -----
 * uplo   specifies whether the matrix data is stored in the upper or the 
 *        lower triangular part of array A. If uplo = 'U' or 'u', then only 
 *        the upper triangular part of A may be referenced. If uplo = 'L' or 
 *        'l', then only the lower triangular part of A may be referenced.
 * trans  specifies op(A). If transa = 'n' or 'N', op(A) = A. If transa = 't',
 *        'T', 'c', or 'C', op(A) = transpose(A)
 * diag   specifies whether or not A is a unit triangular matrix like so:
 *        if diag = 'U' or 'u', A is assumed to be unit triangular. If 
 *        diag = 'N' or 'n', then A is not assumed to be unit triangular.
 * n      specifies the number of rows and columns of the matrix A. It
 *        must be at least 0. In the current implementation n must be <=
 *        4070.
 * A      is a single precision array of dimensions (lda, n). If uplo = 'U' 
 *        or 'u', then A must contains the upper triangular part of a symmetric
 *        matrix, and the strictly lower triangular parts is not referenced. 
 *        If uplo = 'L' or 'l', then A contains the lower triangular part of 
 *        a symmetric matrix, and the strictly upper triangular part is not 
 *        referenced. 
 * lda    is the leading dimension of the two-dimensional array containing A.
 *        lda must be at least max(1, n).
 * x      single precision array of length at least (1 + (n - 1) * abs(incx)).
 *        On entry, x contains the n element right-hand side vector b. On exit,
 *        it is overwritten with the solution vector x.
 * incx   specifies the storage spacing between elements of x. incx must not 
 *        be zero.
 *
 * Output
 * ------
 * x      updated to contain the solution vector x that solves op(A) * x = b.
 *
 * Reference: http://www.netlib.org/blas/strsv.f
 *
 * Error status for this function can be retrieved via cublasGetError().
 *
 * Error Status
 * ------------
 * HIPBLAS_STATUS_NOT_INITIALIZED  if CUBLAS library has not been initialized
 * HIPBLAS_STATUS_INVALID_VALUE    if incx == 0 or if n < 0 or n > 4070
 * HIPBLAS_STATUS_EXECUTION_FAILED if function failed to launch on GPU
 */
__host__ void CUBLASAPI hipblasStrsv (char uplo, char trans, char diag, int n, 
                                     const float *A, int lda, float *x, 
                                     int incx)
{
    struct cublasContext *ctx = CUBLAS_GET_CTX();
    struct cublasStrsvParams params;
    hipError_t cudaStat;
    int info = 0;

    if (!cublasInitialized (ctx)) {
        cublasSetError (ctx, HIPBLAS_STATUS_NOT_INITIALIZED);
        return;
    }

    /* check other inputs */
    if ((toupper (uplo) != 'U') &&
        (toupper (uplo) != 'L')) {
        info = 1;
    }
    else if ((toupper (trans) != 'N') && 
             (toupper (trans) != 'T') && 
             (toupper (trans) != 'C')) {
        info = 2;
    } 
    else if ((toupper (diag) != 'U') &&
             (toupper (diag) != 'N')) {
        info = 3;
    }
    /* FIXME: There should be no upper limit on n */
    else if ((n < 0) || (n > CUBLAS_STRSV_MAX_DIM)) {
        info = 4;
    }
    else if (lda < imax (1, n)) {
        info = 6;
    }
    else if (incx == 0) {
        info = 8;
    }
    if (info) {
        cublasXerbla ("STRSV ", info);
        cublasSetError (ctx, HIPBLAS_STATUS_INVALID_VALUE);
        return;
    }

    /* early out if nothing to do */
    if (n == 0) {
        return;
    }

    memset (&params, 0, sizeof(params));
    params.n = n;
    params.A = A;
    params.lda = lda;
    params.x = x;
    params.incx = incx;
    params.trans = ((toupper(trans) == 'T') || (toupper(trans) == 'C'));
    params.unit = (toupper(diag) == 'U');
    params.up = (toupper(uplo) == 'U');

    cudaStat = hipGetLastError(); /* clear error status */
    if (params.trans) {
        if (params.up) {
            strsv_main_tr_up<<<CUBLAS_STRSV_CTAS,CUBLAS_STRSV_THREAD_COUNT>>>(params);
        } else {
            strsv_main_tr_lo<<<CUBLAS_STRSV_CTAS,CUBLAS_STRSV_THREAD_COUNT>>>(params);
        }
    } else {
        if (params.up) {
            strsv_main_nt_up<<<CUBLAS_STRSV_CTAS,CUBLAS_STRSV_THREAD_COUNT>>>(params);
        } else {
            strsv_main_nt_lo<<<CUBLAS_STRSV_CTAS,CUBLAS_STRSV_THREAD_COUNT>>>(params);
        }
    }        
    cudaStat = hipGetLastError(); /* check for launch error */
    if (cudaStat != hipSuccess) {
        cublasSetError (ctx, HIPBLAS_STATUS_EXECUTION_FAILED);
    }
}

/* column-major ordering */
#define IDXA(row,col)       (parms.lda*(col)+(row))
#define IDXX(i)             (startx + ((i) * parms.incx))
#define XINC                (CUBLAS_STRSV_THREAD_COUNT)

__shared__ float XX[CUBLAS_STRSV_MAX_DIM];  /* cached portion of vector x */
__shared__ float temp;                      /* current diagonal element */

__global__ void strsv_main_tr_up (struct cublasStrsvParams parms) 
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    1
#include "strsv.h"
}
__global__ void strsv_main_tr_lo (struct cublasStrsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 1
#define UP    0
#include "strsv.h"
}

__global__ void strsv_main_nt_up (struct cublasStrsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    1
#include "strsv.h"
}

__global__ void strsv_main_nt_lo (struct cublasStrsvParams parms)
{
#undef  TRANS
#undef  UP
#define TRANS 0
#define UP    0
#include "strsv.h"
}
